#include "hip/hip_runtime.h"
/*******************************************************************************
PASC INFERENCE library
Lukas Pospisil, Illia Horenko, Patrick Gagliardini, Will Sawyer
USI Lugano, 2016
lukas.pospisil@usi.ch

*******************************************************************************/

#include "common.h"
#include "problem.h"
#include "gamma.h"
#include "theta.h"
#include "savevtk.h"
#include "qpsolver.h"

/* PROBLEM SETTINGS */
#define DEFAULT_T 500 /* default length of generated time serie */
#define DEFAULT_K 3 /* default number of clusters */

#define DEBUG_PRINTDATA false /* print values of all data */

#define ALGORITHM_deltaL_eps 0.0001 /*stopping criteria of outer main loop */
#define ALGORITHM_max_s_steps 100 /* max number of outer steps */
#define ALGORITHM_EPSSQUARE 10.0 /* FEM regularization parameter */
#define DEBUG_ALGORITHM_PRINTDATA false /* print values of Theta, Gamma, QPSolver during main cycle */
#define DEBUG_ALGORITHM_PRINTDATA_L true /* print descent of object function in main outer loop */
#define DEBUG_ALGORITHM_PRINTDATA_GAMMA false /* print values of Gamma during main cycle */
#define DEBUG_ALGORITHM_PRINTDATA_THETA false /* print values of Theta during main cycle */

int main( int argc, char *argv[] )
{
	/* parameters of application */
	int dataT = DEFAULT_T; // TODO: do it in a different way
	int gammaK = DEFAULT_K;
	int max_s_steps = ALGORITHM_max_s_steps;

	Initialize(argc,argv); // TODO: load parameters of problem from console input

	/* variables */
	Data data;
	Gamma gamma;
	Theta theta;
	QPSolver qpsolver(&data,&gamma,&theta, ALGORITHM_EPSSQUARE);

	int s; /* index of main iterations */
	Scalar L, L_old, Lgamma, Ltheta, deltaL; /* object function value */

	/* say hello */	
	Message("- start program");
	
	/* generate problem */
	generate_problem(&data,dataT);
	/* print problem */
	if(DEBUG_PRINTDATA){
		data.print();
	}	

	/* initialize gamma */
	gamma.init(data, gammaK);

	/* prepare gammas */
	gamma.prepare_random();	
	if(DEBUG_PRINTDATA){ /* print gamma */
		gamma.print();
	}

	/* initialize theta */
	theta.init(data,gamma);
	if(DEBUG_PRINTDATA){ /* print theta */
		theta.print();
	}

	/* initialize QP solvers */
	qpsolver.init();
	if(DEBUG_PRINTDATA){ /* print state of qpsolver */
		qpsolver.print();
	}
	
	
	/* initialize value of object function */
	L = std::numeric_limits<Scalar>::max(); // TODO: the computation of L should be done in the different way
	
	/* main cycle */
	Message("- run main cycle:");
	for(s=0;s < max_s_steps;s++){
		Message_info_value(" - s = ",s);

		/* --- COMPUTE Theta --- */
		theta.compute(data,gamma);
		if(DEBUG_ALGORITHM_PRINTDATA_THETA || DEBUG_ALGORITHM_PRINTDATA){ /* print theta */
			theta.print(2);
		}
		qpsolver.compute_b();
		Ltheta = qpsolver.get_function_value();
		
		/* --- COMPUTE gamma --- */
		gamma.compute(&qpsolver,data,theta);
		if(DEBUG_ALGORITHM_PRINTDATA_GAMMA || DEBUG_ALGORITHM_PRINTDATA){
			qpsolver.print(2,false);
		}
		qpsolver.compute_b();
		Lgamma = qpsolver.get_function_value();

		/* compute stopping criteria */
		L_old = L;
		L = qpsolver.get_function_value();
		deltaL = abs(L - L_old);

		/* print info about cost function */
		if(DEBUG_ALGORITHM_PRINTDATA_L || DEBUG_ALGORITHM_PRINTDATA){
			Message_info_value("  - L_old       = ",L_old);
			Message_info_value("  - Ltheta      = ",Ltheta);
			Message_info_value("  - Lgamma      = ",Lgamma);
			Message_info_value("  - L           = ",L);
			Message_info_value("  - |L - L_old| = ",deltaL);
		}	

		/* end the main cycle if the change of function value is sufficient */
		if (deltaL < ALGORITHM_deltaL_eps){
			break;
		}
		
	}
	Message("- main cycle finished");

	/* save the solution to VTK */
	if(EXPORT_SAVEVTK){
		Message("- save solution to VTK");
		save_VTK(data,gamma);
	}

	theta.finalize();
	gamma.finalize();
	data.finalize();
	qpsolver.finalize();
	
	/* print info about elapsed time and solution */
	Message_info("- final info:");
	Message_info_time(" - time for computation: ",timer.stop());
	Message_info_value(" - number of iterations: ",s);
	Message_info_value(" - |L - L_old| = ",deltaL);

	/* say bye */	
	Message("- end program");
	
	Finalize();
	return 0;
}

