#include "hip/hip_runtime.h"
/*******************************************************************************
PASC INFERENCE library
Lukas Pospisil, Illia Horenko, Patrick Gagliardini, Will Sawyer
USI Lugano, 2016
lukas.pospisil@usi.ch

*******************************************************************************/

#include "common.h"
#include "problem.h"
#include "gamma.h"
#include "theta.h"
#include "savevtk.h"
#include "qpsolver.h"

/* PROBLEM SETTINGS */
#define DEFAULT_T 10 /* default length of generated time serie */
#define DEFAULT_K 3 /* default number of clusters */

#define DEBUG_PRINTDATA false /* print values of all data */

#define ALGORITHM_deltaL_eps 0.0001 /*stopping criteria of outer main loop */
#define ALGORITHM_max_s_steps 100 /* max number of outer steps */
#define ALGORITHM_EPSSQUARE 10.0 /* FEM regularization parameter */
#define DEBUG_ALGORITHM_PRINTDATA false /* print values of Theta, Gamma, QPSolver during main cycle */
#define DEBUG_ALGORITHM_PRINTDATA_L true /* print descent of object function in main outer loop */
#define DEBUG_ALGORITHM_PRINTDATA_QPIT false /* print number of QPSolver iteration in every outer step */
#define DEBUG_ALGORITHM_PRINTDATA_GAMMA false /* print values of Gamma during main cycle */
#define DEBUG_ALGORITHM_PRINTDATA_THETA false /* print values of Theta during main cycle */

int main( int argc, char *argv[] )
{
	/* parameters of application */
	int dataT = DEFAULT_T; // TODO: do it in a different way
	int gammaK = DEFAULT_K;
	int max_s_steps = ALGORITHM_max_s_steps;

	Initialize(argc,argv); // TODO: load parameters of problem from console input

	/* variables */
	Data data;
	Gamma gamma;
	Theta theta;
	QPSolver qpsolver(&data,&gamma,&theta, ALGORITHM_EPSSQUARE);

	int s; /* index of main iterations */
	Scalar L, L_old, deltaL; /* object function value */


	/* say hello */	
	timer.start(); /* start timer for whole program */
	Message("- start program");
	
	/* generate problem */
	timer.start(); /* start timer for generating problem */
	generate_problem(&data,dataT);
	Message_info_time(" - problem generated in: ",timer.stop());
	
	/* print problem */
	if(DEBUG_PRINTDATA){
		data.print();
	}	

	/* initialize gamma */
	timer.start(); /* start timer for initializing gamma */
	gamma.init(data, gammaK);

	/* prepare gammas */
	gamma.prepare_random();	
	if(DEBUG_PRINTDATA){ /* print gamma */
		gamma.print();
	}
	Message_info_time(" - gamma generated in: ",timer.stop());

	/* initialize theta */
	theta.init(data,gamma);
	if(DEBUG_PRINTDATA){ /* print theta */
		theta.print();
	}

	/* initialize QP solvers */
	qpsolver.init();
	if(DEBUG_PRINTDATA){ /* print state of qpsolver */
		qpsolver.print();
	}

	
	/* initialize value of object function */
	L = std::numeric_limits<Scalar>::max(); // TODO: the computation of L should be done in the different way
	
	/* main cycle */
	Message("- run main cycle:");
	for(s=0;s < max_s_steps;s++){
		Message_info_value(" - s = ",s);

		/* --- COMPUTE Theta --- */
		timer.start(); /* start timer for solving Theta-problem */
		theta.compute(data,gamma);
		if(DEBUG_ALGORITHM_PRINTDATA_THETA || DEBUG_ALGORITHM_PRINTDATA){ /* print theta */
			theta.print(2);
		}
		Message_info_time("  - theta problem solved in: ",timer.stop());

		
		/* --- COMPUTE gamma --- */
		timer.start(); /* start timer for solving gamma-problem */
		gamma.compute(&qpsolver,data,theta);
		if(DEBUG_ALGORITHM_PRINTDATA_GAMMA || DEBUG_ALGORITHM_PRINTDATA){
			qpsolver.print(2);
		}
		Message_info_time("  - gamma problem solved in: ",timer.stop());

		/* compute stopping criteria */
		L_old = L;
		L = qpsolver.get_function_value();
		deltaL = abs(L - L_old);

		/* print info about cost function */
		if(DEBUG_ALGORITHM_PRINTDATA_L || DEBUG_ALGORITHM_PRINTDATA){
//			Message_info_value("  - L_old       = ",L_old);
			Message_info_value("  - L           = ",L);
//			Message_info_value("  - |L - L_old| = ",deltaL);
		}	

		/* end the main cycle if the change of function value is sufficient */
		if (deltaL < ALGORITHM_deltaL_eps){
			break;
		}
		
	}
	Message("- main cycle finished");

	/* save the solution to VTK */
	if(EXPORT_SAVEVTK){
		Message("- save solution to VTK");
		save_VTK(data,gamma);
	}

	theta.finalize();
	gamma.finalize();
	data.finalize();
	qpsolver.finalize();
	
	/* print info about elapsed time and solution */
	Message_info("- final info:");
	Message_info_time(" - time for computation: ",timer.stop());
	Message_info_value(" - number of outer iterations: ",s);
	Message_info_value(" - |L - L_old| = ",deltaL);
	Message_info(" - QPSolver:");
	Message_info_value("  - it =         ", qpsolver.get_it_all());
	Message_info_value("  - hessmult =  ", qpsolver.get_hessmult_all());
	Message_info_time( "  - time =       ", qpsolver.get_time_total());
	Message_info_time( "   - t_init =     ", qpsolver.get_time_init());
	Message_info_time( "   - t_project =  ", qpsolver.get_time_projection());
	Message_info_time( "   - t_matmult =  ", qpsolver.get_time_matmult());
	Message_info_time( "   - t_dot =      ", qpsolver.get_time_dot());
	Message_info_time( "   - t_update =   ", qpsolver.get_time_update());
	Message_info_time( "   - t_other =    ", qpsolver.get_time_other());

	/* say bye */	
	Message("- end program");
	Message_info_value("- timer status: ",timer.status());

	
	Finalize();
	return 0;
}

