#include "hip/hip_runtime.h"
/*******************************************************************************
PASC INFERENCE library
Lukas Pospisil, Illia Horenko, Patrick Gagliardini, Will Sawyer
USI Lugano, 2016
lukas.pospisil@usi.ch

*******************************************************************************/

#include "common.h"
#include "problem.h"
#include "gamma.h"
#include "theta.h"
#include "savevtk.h"

/* include QPSolver */
//#include "qpsolver_projectionstep.h"


int main( int argc, char *argv[] )
{
	/* parameters of application */
	int dataT = DEFAULT_T; // TODO: do it in a different way
	int gammaK = DEFAULT_K;
	int max_s_steps = ALGORITHM_max_s_steps;

	Initialize(argc,argv); // TODO: load parameters of problem from console input

	/* variables */
	Data data;
	Gamma gamma;
	Theta theta;
	int s; /* index of main iterations */
	Scalar L, L_old, deltaL; /* object function value */

	/* say hello */	
	Message("- start program");
	
	/* generate problem */
	generate_problem(&data,dataT);
	/* print problem */
	if(DEBUG_PRINTDATA){
		data.print();
	}	

	/* initialize gamma */
	gamma.init(data, gammaK);

	/* prepare gammas */
	gamma.prepare_random();	
	if(DEBUG_PRINTDATA){ /* print gamma */
		gamma.print();
	}

	/* initialize theta */
	theta.init(data,gamma);
	if(DEBUG_PRINTDATA){ /* print theta */
		theta.print();
	}

	/* initialize QP solvers */
//	QPSolver qpsolver;
//	QPSolver *qpsolverpermon = new QPSolverPermon(&data, &gamma, &theta, eps_sqr);
//	QPSolver *qpsolverprojectionstep = new QPSolverProjectionstep(&data, &gamma, &theta, eps_sqr);
//	qpsolverpermon->init();
//	qpsolverprojectionstep->init();
//	qpsolver = qpsolverprojectionstep;
	
	/* initialize value of object function */
	L = std::numeric_limits<Scalar>::max(); // TODO: the computation of L should be done in the different way
	
	/* main cycle */
	Message("- run main cycle:");
	for(s=0;s < max_s_steps;s++){
		Message_info_value(" - s = ",s);

		/* --- COMPUTE Theta --- */
		theta.compute(data,gamma);
		if(DEBUG_PRINTDATA){ /* print theta */
			theta.print(2);
		}
		
		/* --- COMPUTE gamma --- */
/*		gamma.compute(qpsolver,data,theta); CHKERRQ(ierr);
		if(DEBUG_PRINTDATA){
			ierr = qpsolver->print(my_viewer); CHKERRQ(ierr);
		}
*/
		/* compute stopping criteria */
		L_old = L;
//		ierr = qpsolver->get_function_value(&L); CHKERRQ(ierr);
//		deltaL = PetscAbsScalar(L - L_old);

		/* print info about cost function */
		if(DEBUG_PRINTL){
			Message_info_value("  - L_old       = ",L_old);
			Message_info_value("  - L           = ",L);
			Message_info_value("  - |L - L_old| = ",deltaL);
		}	

		/* end the main cycle if the change of function value is sufficient */
		if (deltaL < ALGORITHM_deltaL_eps){
			break;
		}
		
	}
	Message("- main cycle finished");

	/* save the solution to VTK */
	if(EXPORT_SAVEVTK){
		Message("- save solution to VTK");
		save_VTK(data,gamma);
	}

	theta.finalize();
	gamma.finalize();
	data.finalize();

//	qpsolverprojectionstep->finalize();

	
	/* print info about elapsed time and solution */

	Message_info("- final info:");
	Message_info_time(" - time for computation: ",timer.stop());
	Message_info_value(" - number of iterations: ",s);
	Message_info_value(" - |L - L_old| = ",deltaL);

	/* say bye */	
	Message("- end program");
	
	Finalize();
	return 0;
}

