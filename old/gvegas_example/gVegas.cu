#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#include <cutil_inline.h>

#include "vegas.h" 
#include "vegasconst.h"
#include "kernels.h"

#include "gvegas.h"

#include "getrusage_sec.h"

void gVegas(double& avgi, double& sd, double& chi2a)
{

   for (int j=0;j<ndim;j++) {
      xi[j][0] = 1.;
   }

   // entry vegas1

   it = 0;

   // entry vegas2
   nd = nd_max;
   ng = 1;
   
   npg = 0;
   if (mds!=0) {
      
      ng = (int)pow((0.5*(double)ncall),1./(double)ndim);
      mds = 1;
      if (2*ng>=nd_max) {
         mds = -1;
         npg = ng/nd_max+1;
         nd = ng/npg;
         ng = npg*nd;
      }
      
   }
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_ndim), &ndim, sizeof(int)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_ng),   &ng,   sizeof(int)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_nd),   &nd,   sizeof(int)));
   hipDeviceSynchronize(); // wait for synchronize

   nCubes = (unsigned)(pow(ng,ndim));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_nCubes), &nCubes, sizeof(nCubes)));
   hipDeviceSynchronize(); // wait for synchronize

   npg = ncall/nCubes;
   if (npg<2) npg = 2;
   calls = (double)(npg*nCubes);

   unsigned nCubeNpg = nCubes*npg;

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << vegas internal parameters >>"<<std::endl;
      std::cout<<"            ng: "<<std::setw(5)<<ng<<std::endl;
      std::cout<<"            nd: "<<std::setw(5)<<nd<<std::endl;
      std::cout<<"           npg: "<<std::setw(5)<<npg<<std::endl;
      std::cout<<"        nCubes: "<<std::setw(12)<<nCubes<<std::endl;
      std::cout<<"    nCubes*npg: "<<std::setw(12)<<nCubeNpg<<std::endl;
   }
   
   dxg = 1./(double)ng;
   double dnpg = (double)npg;
   double dv2g = calls*calls*pow(dxg,ndim)*pow(dxg,ndim)/(dnpg*dnpg*(dnpg-1.));
   xnd = (double)nd;
   dxg *= xnd;
   xjac = 1./(double)calls;
   for (int j=0;j<ndim;j++) {
      dx[j] = xu[j]-xl[j];
      xjac *= dx[j];
   }

   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_npg),  &npg,  sizeof(int)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xjac), &xjac, sizeof(double)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_dxg),  &dxg,  sizeof(double)));
   hipDeviceSynchronize(); // wait for synchronize

   ndo = 1;

   if (nd!=ndo) {

      double rc = (double)ndo/xnd;
      
      for (int j=0;j<ndim;j++) {
         
         int k = -1;
         double xn = 0.;
         double dr = 0.;
         int i = k;
         k++;
         dr += 1.;
         double xo = xn;
         xn = xi[j][k];

         while (i<nd-1) {

            while (dr<=rc) {
               k++;
               dr += 1.;
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn - (xn-xo)*dr;
         }
         
         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (double)xin[i];
         }
         xi[j][nd-1] = 1.;

      }
      ndo = nd;
      
   }

   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xl), xl, sizeof(xl)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_dx), dx, sizeof(dx)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
   hipDeviceSynchronize(); // wait for synchronize

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << input parameters for vegas >>"<<std::endl;
      std::cout<<"     ndim ="<<std::setw(3)<<ndim
               <<"   ncall ="<<std::setw(10)<<(int)calls<<std::endl;
      std::cout<<"     it   =  0"
               <<"   itmx ="<<std::setw(5)<<itmx<<std::endl;
      std::cout<<"     acc  = "<<std::fixed
               <<std::setw(9)<<std::setprecision(3)<<acc<<std::endl;
      std::cout<<"     mds  ="<<std::setw(3)<<mds
               <<"   nd = "<<std::setw(4)<<nd<<std::endl;
      for (int j=0;j<ndim;j++) {
         std::cout<<"    (xl,xu)= ( "<<std::setw(6)<<std::fixed
                  <<xl[j]<<" , "<<xu[j]<<" )"<<std::endl;
      }
      
   }

   // entry vegas3

   it = 0;
   si = 0.;
   si2 = 0.;
   swgt = 0.;
   schi = 0.;

   //--------------------------
   //  Set up kernel vaiables
   //--------------------------
   const int nGridSizeMax =  65535;
   
   dim3 ThBk(nBlockSize);

   int nGridSizeX, nGridSizeY;
   int nBlockTot = (nCubeNpg-1)/nBlockSize+1;
   nGridSizeY = (nBlockTot-1)/nGridSizeMax+1;
   nGridSizeX = (nBlockTot-1)/nGridSizeY+1;
   dim3 BkGd(nGridSizeX, nGridSizeY);

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << kernel parameters for CUDA >>"<<std::endl;
      std::cout<<"       Block size           ="<<std::setw(7)<<ThBk.x<<std::endl;
      std::cout<<"       Grid size            ="<<std::setw(7)<<BkGd.x
               <<" x "<<BkGd.y<<std::endl;
      int nThreadsTot = ThBk.x*BkGd.x*BkGd.y;
      std::cout<<"     Actual Number of calls ="<<std::setw(12)
               <<nThreadsTot<<std::endl;
      std::cout<<"   Required Number of calls ="<<std::setw(12)
               <<nCubeNpg<<" ( "<<std::setw(6)<<std::setprecision(2)
               <<100.*(double)nCubeNpg/(double)nThreadsTot<<"%)"<<std::endl;
      std::cout<<std::endl;
   }
      
   // allocate Fval
   int sizeFval = nCubeNpg*sizeof(double);

   // CPU
   double* hFval;
   cutilSafeCall(hipHostMalloc((void**)&hFval, sizeFval));
   memset(hFval, '\0', sizeFval);

   // GPU
   double* gFval;
   cutilSafeCall(hipMalloc((void**)&gFval, sizeFval));

   // allocate IAval
   int sizeIAval = nCubeNpg*ndim*sizeof(int);

   // CPU
   int* hIAval;
   cutilSafeCall(hipHostMalloc((void**)&hIAval, sizeIAval));
   memset(hIAval, '\0', sizeIAval);

   // GPU
   int* gIAval;
   cutilSafeCall(hipMalloc((void**)&gIAval, sizeIAval));

   double startVegasCall, endVegasCall;
   double startVegasMove, endVegasMove;
   double startVegasFill, endVegasFill;
   double startVegasRefine, endVegasRefine;

   do {
      
      it++;

      startVegasCall = getrusage_usec();
      gVegasCallFunc<<<BkGd, ThBk>>>(gFval, gIAval);
      hipDeviceSynchronize(); // wait for synchronize
      endVegasCall = getrusage_usec();
      timeVegasCall += endVegasCall-startVegasCall;

      startVegasMove = getrusage_usec();
      cutilSafeCall(hipMemcpy(hFval, gFval,  sizeFval,
                               hipMemcpyDeviceToHost));

      cutilSafeCall(hipMemcpy(hIAval, gIAval,  sizeIAval,
                               hipMemcpyDeviceToHost));
      endVegasMove = getrusage_usec();
      timeVegasMove += endVegasMove-startVegasMove;

// *****************         

      startVegasFill = getrusage_usec();

      ti = 0.;
      tsi = 0.;

      double d[ndim_max][nd_max];

      for (int j=0;j<ndim;++j) {
         for (int i=0;i<nd;++i) {
            d[j][i] = 0.;
         }
      }

      for (unsigned ig=0;ig<nCubes;ig++) {
         double fb = 0.;
         double f2b = 0.;
         for (int ipg=0;ipg<npg;ipg++) {
            int idx = npg*ig+ipg;
            double f = hFval[idx];
            double f2 = f*f;
            fb += f;
            f2b += f2;
         }
         f2b = sqrt(f2b*npg);
         f2b = (f2b-fb)*(f2b+fb);
         ti += fb;
         tsi += f2b;
         if (mds<0) {
            int idx = npg*ig;
            for (int idim=0;idim<ndim;idim++) {
               int iaj = hIAval[idim*nCubeNpg+idx];
               d[idim][iaj] += f2b;
            }
         }
      }

      if (mds>0) {
         for (int idim=0;idim<ndim;idim++) {
            int idimCube = idim*nCubeNpg;
            for (int idx=0;idx<nCubeNpg;idx++) {
               double f = hFval[idx];
               int iaj = hIAval[idimCube+idx];
               d[idim][iaj] += f*f;
            }
         }
      }

      endVegasFill = getrusage_usec();
      timeVegasFill += endVegasFill-startVegasFill;

      tsi *= dv2g;
      double ti2 = ti*ti;
      double wgt = ti2/tsi;
      si += ti*wgt;
      si2 += ti2;
      swgt += wgt;
      schi += ti2*wgt;
      avgi = si/swgt;
      sd = swgt*it/si2;
      chi2a = 0.;
      if (it>1) chi2a = sd*(schi/swgt-avgi*avgi)/((double)it-1.);
      sd = sqrt(1./sd);
      
      if (nprn!=0) {
         tsi = sqrt(tsi);
         std::cout<<std::endl;
         std::cout<<" << integration by vegas >>"<<std::endl;
         std::cout<<"     iteration no. "<<std::setw(4)<<it
                  <<std::setw(10)<<std::setprecision(6)
                  <<"   integral=  "<<ti<<std::endl;
         std::cout<<"                          std dev  = "<<tsi<<std::endl;
         std::cout<<"     accumulated results: integral = "<<avgi<<std::endl;
         std::cout<<"                          std dev  = "<<sd<<std::endl;
	 if (it > 1) {
            std::cout<<"                          chi**2 per it'n = "
                     <<std::setw(10)<<std::setprecision(4)<<chi2a<<std::endl;
         }
         if (nprn<0) {
            for (int j=0;j<ndim;j++) {
               std::cout<<"   == data for axis "
                        <<std::setw(2)<<j<<" --"<<std::endl;
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce"<<std::endl;
            }
         }
      }

      // refine grid

      startVegasRefine = getrusage_usec();
      
      double r[nd_max];
      double dt[ndim_max];
      for (int j=0;j<ndim;j++) {
         double xo = d[j][0];
         double xn = d[j][1];
         d[j][0] = 0.5*(xo+xn);
         dt[j] = d[j][0];
         for (int i=1;i<nd-1;i++) {
            d[j][i] = xo+xn;
            xo = xn;
            xn = d[j][i+1];
            d[j][i] = (d[j][i]+xn)/3.;
            dt[j] += d[j][i];
         }
         d[j][nd-1] = 0.5*(xn+xo);
         dt[j] += d[j][nd-1];
      }
      
      for (int j=0;j<ndim;j++) {
         double rc = 0.;
         for (int i=0;i<nd;i++) {
            r[i] = 0.;
            if (d[j][i]>0.) {
               double xo = dt[j]/d[j][i];
               if (!isinf(xo)) 
                  r[i] = pow(((xo-1.)/xo/log(xo)),alph);
            }
            rc += r[i];
         }
         rc /= xnd;
         int k = -1;
         double xn = 0.;
         double dr = xn;
         int i = k;
         k++;
         dr += r[k];
         double xo = xn;
         xn = xi[j][k];
         
         do {

            while (dr<=rc) {
               k++;
               dr += r[k];
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn-(xn-xo)*dr/r[k];

         } while (i<nd-2);

         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (double)xin[i];
         }
         xi[j][nd-1] = 1.;

      }
      cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
      hipDeviceSynchronize(); // wait for synchronize

      endVegasRefine = getrusage_usec();
      timeVegasRefine += endVegasRefine-startVegasRefine;
      
   } while (it<itmx && acc*fabs(avgi)<sd);


   cutilSafeCall(hipHostFree(hFval));
   cutilSafeCall(hipFree(gFval));

   cutilSafeCall(hipHostFree(hIAval));
   cutilSafeCall(hipFree(gIAval));

}
