#include "hip/hip_runtime.h"
#include "external/petscvector/common/fem.h"

namespace pascinference {
namespace common {

void Fem<PetscVector>::ExternalContent::cuda_Fem_cuda_occupancy(){
	LOG_FUNC_BEGIN

	/* compute optimal kernel calls */
	gpuErrchk( hipOccupancyMaxPotentialBlockSize( &minGridSize_reduce, &blockSize_reduce, kernel_fem_reduce_data, 0, 0) );
	gridSize_reduce = (decomposition2->get_Tlocal() + blockSize_reduce - 1)/ blockSize_reduce;

	gpuErrchk( hipOccupancyMaxPotentialBlockSize( &minGridSize_prolongate, &blockSize_prolongate, kernel_fem_prolongate_data, 0, 0) );
	gridSize_prolongate = (decomposition2->get_Tlocal() + blockSize_prolongate - 1)/ blockSize_prolongate;

	LOG_FUNC_END
}

__global__ void kernel_Fem_reduce_data(double *data1, double *data2, int T1, int T2, int T2local, double diff) {
	int t2 = blockIdx.x*blockDim.x + threadIdx.x;

	if(t2 < T2local){
		double mysum = 0.0;
		for(int i=round(t2*diff); i < round((t2+1)*diff);i++){
			mysum += data1[i];
		}

		data2[t2] = mysum;
	}
}


__global__ void kernel_Fem_prolongate_data(double *data1, double *data2, int T1, int T2, int T2local, double diff) {
	int t2 = blockIdx.x*blockDim.x + threadIdx.x;

	if(t2 < T2local){
		for(int i=round(t2*diff); i < round((t2+1)*diff);i++){
			data1[i] = data2[t2];
		}
	}
}


}
} /* end of namespace */
