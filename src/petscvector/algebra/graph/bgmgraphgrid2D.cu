#include "external/petscvector/algebra/graph/bgmgraphgrid2D.h"

namespace pascinference {
namespace algebra {

void BGMGraphGrid2D<PetscVector>::ExternalContent::process_grid_cuda(){
	LOG_FUNC_BEGIN

	/* copy data to gpu */
	gpuErrchk( hipMalloc((void **)&neighbor_nmbs_gpu, n*sizeof(int)) );	
	gpuErrchk( hipMemcpy( neighbor_nmbs_gpu, neighbor_nmbs, n*sizeof(int), hipMemcpyHostToDevice) );

	/* allocate pointers on CPU */
	neighbor_ids_cpugpu = (int**)malloc(n*sizeof(int*));
		
	for(int i=0;i<n;i++){
		int mysize = neighbor_nmbs[i];

		gpuErrchk( hipMalloc((void **)&(neighbor_ids_cpugpu[i]), mysize*sizeof(int)) );
		gpuErrchk( hipMemcpy( neighbor_ids_cpugpu[i], neighbor_ids[i], mysize*sizeof(int), hipMemcpyHostToDevice) );
	}

	/* copy pointers to arrays from CPU to GPU */
	gpuErrchk( hipMalloc((void **)&neighbor_ids_gpu, n*sizeof(int*)) );
	gpuErrchk( hipMemcpy( neighbor_ids_gpu, neighbor_ids_cpugpu, n*sizeof(int*), hipMemcpyHostToDevice) );

	gpuErrchk( hipDeviceSynchronize() );
	
	LOG_FUNC_END
}


}
} /* end of namespace */
