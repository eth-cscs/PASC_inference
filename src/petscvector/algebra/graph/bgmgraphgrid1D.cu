#include "external/petscvector/petscvector.cuh"
#include "external/petscvector/algebra/graph/bgmgraphgrid1D.h"

namespace pascinference {
namespace algebra {

void BGMGraphGrid1D<PetscVector>::ExternalContent::process_grid_cuda(int *neighbor_nmbs, int **neighbor_ids){
	/* copy data to gpu */
	gpuErrchk( hipMalloc((void **)&neighbor_nmbs_gpu, n*sizeof(int)) );	
	gpuErrchk( hipMemcpy( neighbor_nmbs_gpu, neighbor_nmbs, n*sizeof(int), hipMemcpyHostToDevice) );

	/* allocate pointers on CPU */
	neighbor_ids_cpugpu = (int**)malloc(n*sizeof(int*));
		
	for(int i=0;i<n;i++){
		int mysize = neighbor_nmbs[i];

		gpuErrchk( hipMalloc((void **)&(neighbor_ids_cpugpu[i]), mysize*sizeof(int)) );
		gpuErrchk( hipMemcpy( neighbor_ids_cpugpu[i], neighbor_ids[i], mysize*sizeof(int), hipMemcpyHostToDevice) );
	}

	/* copy pointers to arrays from CPU to GPU */
	gpuErrchk( hipMalloc((void **)&neighbor_ids_gpu, n*sizeof(int*)) );
	gpuErrchk( hipMemcpy( neighbor_ids_gpu, neighbor_ids_cpugpu, n*sizeof(int*), hipMemcpyHostToDevice) );

	gpuErrchk( hipDeviceSynchronize() );
}




}
} /* end of namespace */

