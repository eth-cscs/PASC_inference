#include "external/petscvector/petscvector.cuh"
#include "external/petscvector/algebra/graph/bgmgraph.h"

namespace pascinference {
namespace algebra {
	
void BGMGraph<PetscVector>::ExternalContent::cuda_destroy(){
	LOG_FUNC_BEGIN

	gpuErrchk( hipFree(neighbor_nmbs_gpu) );
	for(int i=0;i<n;i++){
		gpuErrchk( hipFree(neighbor_ids_cpugpu[i]) );
	}
	free(neighbor_ids_cpugpu);
	gpuErrchk( hipFree(neighbor_ids_gpu) );

	LOG_FUNC_END
}

void BGMGraph<PetscVector>::ExternalContent::cuda_process(int *neighbor_nmbs, int **neighbor_ids){
	LOG_FUNC_BEGIN

	/* copy data to gpu */
	gpuErrchk( hipMalloc((void **)&neighbor_nmbs_gpu, n*sizeof(int)) );
	gpuErrchk( hipMemcpy( neighbor_nmbs_gpu, neighbor_nmbs, n*sizeof(int), hipMemcpyHostToDevice) );
		
	/* allocate pointers on CPU */
	neighbor_ids_cpugpu = (int**)malloc(n*sizeof(int*));
		
	for(int i=0;i<n;i++){
		int mysize = neighbor_nmbs[i];
		
		gpuErrchk( hipMalloc((void **)&(neighbor_ids_cpugpu[i]), mysize*sizeof(int)) );
		gpuErrchk( hipMemcpy( neighbor_ids_cpugpu[i], neighbor_ids[i], mysize*sizeof(int), hipMemcpyHostToDevice) );
	}

	/* copy pointers to arrays from CPU to GPU */
	gpuErrchk( hipMalloc((void **)&neighbor_ids_gpu, n*sizeof(int*)) );
	gpuErrchk( hipMemcpy( neighbor_ids_gpu, neighbor_ids_cpugpu, n*sizeof(int*), hipMemcpyHostToDevice) );

	gpuErrchk( hipDeviceSynchronize() );

	LOG_FUNC_END
}

}
}
