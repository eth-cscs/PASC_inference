#include "hip/hip_runtime.h"
#include "external/petscvector/petscvector.cuh"
#include "external/petscvector/algebra/integration/entropyintegrationcudavegas.h"

namespace pascinference {
namespace algebra {

const int ndim_max = 10;
const int nd_max = 50;

__device__ __constant__ int g_ndim;
__device__ __constant__ int g_ng;
__device__ __constant__ int g_npg;
__device__ __constant__ int g_nd;
__device__ __constant__ double g_xjac;
__device__ __constant__ double g_dxg;
__device__ __constant__ double g_xl[ndim_max];
__device__ __constant__ double g_dx[ndim_max];
__device__ __constant__ double g_xi[ndim_max][nd_max];
__device__ __constant__ unsigned g_nCubes;		


EntropyIntegrationCudaVegas<PetscVector>::ExternalContent::ExternalContent() {
	LOG_FUNC_BEGIN

	/* restart timers */
	this->timerVegasCall.restart();
	this->timerVegasMove.restart();
	this->timerVegasFill.restart();
	this->timerVegasRefine.restart();

	LOG_FUNC_END
}

void EntropyIntegrationCudaVegas<PetscVector>::ExternalContent::cuda_gVegas(double &avgi, double &sd, double &chi2a) {
	LOG_FUNC_BEGIN

	int mds = 1;
	int nprn = 1;
	const double alph = 1.5;

	int it;
	int nd;
	int ng;
	int ndo;
	int npg;
	int nCubes;
	double calls;
	double dxg;
	double dnpg;
	double dv2g;
	
	int nGridSizeX, nGridSizeY;
	int nBlockTot;
	
	double xi[ndim_max][nd_max];
	double xl[ndim_max],xu[ndim_max];
	double dx[ndim_max];
	double xin[nd_max];

	double xnd;
	double xjac;
	
	double si;
	double si2;
	double swgt;
	double schi;
	
	for (int i=0;i< this->ndim;i++) {
		xl[i] = 0.;
		xu[i] = 1.;
	}

	for (int j=0; j < this->ndim; j++) {
		xi[j][0] = 1.;
	}

	/* entry vegas1 */
	it = 0;

	/* entry vegas2 */
	nd = nd_max;
	ng = 1;
   
	npg = 0;
	if (mds!=0) {
		ng = (int)pow((0.5*(double)(this->ncall)),1./(double)(this->ndim));
		mds = 1;
		if (2*ng>=nd_max) {
			mds = -1;
			npg = ng/(double)nd_max+1;
			nd = ng/(double)npg;
			ng = npg*nd;
		}
	}

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_ndim), &(this->ndim), sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_ng),   &ng,   sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_nd),   &nd,   sizeof(int)));
	hipDeviceSynchronize(); /* wait for synchronize */

	nCubes = (unsigned)(pow(ng,this->ndim));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_nCubes), &nCubes, sizeof(nCubes)));
	hipDeviceSynchronize(); /* wait for synchronize */

	npg = ncall/(double)nCubes;
	if(npg < 2){
		npg = 2;
	}
	calls = (double)(npg*nCubes);

	unsigned nCubeNpg = nCubes*npg;

	if (nprn!=0) {
		coutMaster << std::endl;
		coutMaster << " << vegas internal parameters >> " << std::endl;
		coutMaster << "            ng: " << std::setw(5) << ng << std::endl;
		coutMaster << "            nd: " << std::setw(5) << nd << std::endl;
		coutMaster << "           npg: " << std::setw(5) << npg << std::endl;
		coutMaster << "        nCubes: " << std::setw(12) << nCubes << std::endl;
		coutMaster << "    nCubes*npg: " << std::setw(12) << nCubeNpg << std::endl;
	}

	dxg = 1./(double)ng;
	dnpg = (double)npg;
	dv2g = calls*calls*pow(dxg,this->ndim)*pow(dxg,this->ndim)/(dnpg*dnpg*(dnpg-1.));
	xnd = (double)nd;
	dxg *= xnd;
	xjac = 1./(double)calls;
	for (int j=0;j<this->ndim;j++) {
		dx[j] = xu[j]-xl[j];
		xjac *= dx[j];
	}

	/* tranfer data to GPU */
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_npg),  &npg,  sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_xjac), &xjac, sizeof(double)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_dxg),  &dxg,  sizeof(double)));
	hipDeviceSynchronize();

	ndo = 1;

	if (nd!=ndo) {
		double rc = (double)ndo/xnd;

		for (int j=0;j<this->ndim;j++) {
			int k = -1;
			double xn = 0.;
			double dr = 0.;
			int i = k;
			k++;
			dr += 1.;
			double xo = xn;
			xn = xi[j][k];

			while (i<nd-1) {
				while (dr<=rc) {
					k++;
					dr += 1.;
					xo = xn;
					xn = xi[j][k];
				}
				i++;
				dr -= rc;
				xin[i] = xn - (xn-xo)*dr;
			}

			for (int i=0;i<nd-1;i++) {
				xi[j][i] = (double)xin[i];
			}
			xi[j][nd-1] = 1.;
		}
		ndo = nd;
	}

	/* transfer data to GPU */
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_xl), xl, sizeof(xl)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_dx), dx, sizeof(dx)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
	hipDeviceSynchronize();
	
	if (nprn!=0) {
		coutMaster << std::endl;
		coutMaster << " << input parameters for vegas >>" << std::endl;
		coutMaster << "     ndim =" << std::setw(3) << this->ndim
					<< "   ncall = " << std::setw(10) << this->ncall <<std::endl;
		coutMaster << "     it   =  0"
					<< "   itmx = " << std::setw(5) << this->itmx << std::endl;
		coutMaster << "     acc  = " << std::fixed
					<< std::setw(9) << std::setprecision(3) << this->acc << std::endl;
		coutMaster << "     mds  = " << std::setw(3) << mds
					<< "   nd = " << std::setw(4) << nd <<std::endl;
		for(int j=0; j < this->ndim; j++){
			coutMaster << "    (xl,xu)= ( " << std::setw(6) << std::fixed
						<< xl[j] << ", " << xu[j] << " )" << std::endl;
		}
	}	

	/* entry vegas3 */
	it = 0;
	si = 0.;
	si2 = 0.;
	swgt = 0.;
	schi = 0.;

	/* --------------------------
	 * Set up kernel vaiables
     * --------------------------
     */
	const int nGridSizeMax =  65535;
   
	dim3 ThBk(nBlockSize);
	
	nBlockTot = (nCubeNpg-1)/nBlockSize+1;
	nGridSizeY = (nBlockTot-1)/nGridSizeMax+1;
	nGridSizeX = (nBlockTot-1)/nGridSizeY+1;
	dim3 BkGd(nGridSizeX, nGridSizeY);

	if (nprn!=0) {
		coutMaster << std::endl;
		coutMaster << " << kernel parameters for CUDA >> " << std::endl;
		coutMaster << "       Block size           = " << std::setw(7) << ThBk.x << std::endl;
		coutMaster << "       Grid size            = " << std::setw(7) << BkGd.x
					<< " x " << BkGd.y << std::endl;
		int nThreadsTot = ThBk.x*BkGd.x*BkGd.y;
		coutMaster << "     Actual Number of calls = " << std::setw(12)
					<< nThreadsTot << std::endl;
		coutMaster << "   Required Number of calls = " << std::setw(12)
					<< nCubeNpg << " ( " << std::setw(6) << std::setprecision(2)
					<< 100.*(double)nCubeNpg/(double)nThreadsTot << "%)" <<std::endl;
		coutMaster << std::endl;
	}

	int sizeFval;
	double* hFval;
	double* gFval;
	
	int sizeIAval;
	int* hIAval;
	int* gIAval;

	double startVegasCall, endVegasCall;
	double startVegasMove, endVegasMove;
	double startVegasFill, endVegasFill;
	double startVegasRefine, endVegasRefine;

	/* allocate Fval */
	sizeFval = nCubeNpg*sizeof(double);

	/* CPU */
	gpuErrchk(hipHostMalloc((void**)&hFval, sizeFval));
	memset(hFval, '\0', sizeFval);

	/* GPU */
	gpuErrchk(hipMalloc((void**)&gFval, sizeFval));

	/* allocate IAval */
	sizeIAval = nCubeNpg*ndim*sizeof(int);

	/* CPU */
	gpuErrchk(hipHostMalloc((void**)&hIAval, sizeIAval));
	memset(hIAval, '\0', sizeIAval);

	/* GPU */
	gpuErrchk(hipMalloc((void**)&gIAval, sizeIAval));

	/* perform main iterations */
	do {
		it++;
		
		/* call integral function */
		timerVegasCall.start();
		gVegasCallFunc<<<BkGd, ThBk>>>(gFval, gIAval);
		hipDeviceSynchronize();
		timerVecgasCall.stop();

		/* move computed results */
		timerVegasMove.start();
		gpuErrchk(hipMemcpy(hFval, gFval,  sizeFval,
                               hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(hIAval, gIAval,  sizeIAval,
                               hipMemcpyDeviceToHost));
		timerVegasMove.stop();

		/* fill */
		timerVegasFill.start();

		ti = 0.;
		tsi = 0.;

		double d[ndim_max][nd_max];

		for (int j=0;j<ndim;++j) {
			for (int i=0;i<nd;++i) {
				d[j][i] = 0.;
			}
		}

		for (unsigned ig=0;ig<nCubes;ig++) {
			double fb = 0.;
			double f2b = 0.;
			for(int ipg=0;ipg<npg;ipg++) {
				int idx = npg*ig+ipg;
				double f = hFval[idx];
				double f2 = f*f;
				fb += f;
				f2b += f2;
			}
			f2b = sqrt(f2b*npg);
			f2b = (f2b-fb)*(f2b+fb);
			ti += fb;
			tsi += f2b;
			if(mds<0){
				int idx = npg*ig;
				for(int idim=0;idim<ndim;idim++) {
					int iaj = hIAval[idim*nCubeNpg+idx];
					d[idim][iaj] += f2b;
				}
			}
		}

		if(mds>0){
			for (int idim=0;idim<ndim;idim++) {
				int idimCube = idim*nCubeNpg;
				for(int idx=0;idx<nCubeNpg;idx++) {
					double f = hFval[idx];
					int iaj = hIAval[idimCube+idx];
					d[idim][iaj] += f*f;
				}
			}
		}

		endVegasFill = getrusage_usec();
		timerVegasFill.stop();

		tsi *= dv2g;
		double ti2 = ti*ti;
		double wgt = ti2/tsi;
		si += ti*wgt;
		si2 += ti2;
		swgt += wgt;
		schi += ti2*wgt;
		avgi = si/swgt;
		sd = swgt*it/si2;
		chi2a = 0.;
		if(it>1) chi2a = sd*(schi/swgt-avgi*avgi)/((double)it-1.);
		sd = sqrt(1./sd);
      
		if(nprn!=0) {
			tsi = sqrt(tsi);
			coutMaster << std::endl;
			coutMaster << " << integration by vegas >>" << std::endl;
			coutMaster << "     iteration no. " << std::setw(4) << it
						<< std::setw(10) << std::setprecision(6)
						<< "   integral=  " << ti << std::endl;
			coutMaster << "                          std dev  = " << tsi << std::endl;
			coutMaster << "     accumulated results: integral = " << avgi << std::endl;
			coutMaster << "                          std dev  = " << sd << std::endl;
			if(it > 1){
				coutMaster << "                          chi**2 per it'n = "
							<< std::setw(10) << std::setprecision(4) << chi2a << std::endl;
			}
			if(nprn<0){
				for (int j=0;j<ndim;j++) {
					coutMaster << "   == data for axis "
								<< std::setw(2) << j << " --" << std::endl;
					coutMaster << "    x    delt i   convce";
					coutMaster << "    x    delt i   convce";
					coutMaster << "    x    delt i   convce"<<std::endl;
				}
			}
		}

		/* refine grid */
		timerVegasRefine.start();

		double r[nd_max];
		double dt[ndim_max];
		for(int j=0;j<ndim;j++) {
			double xo = d[j][0];
			double xn = d[j][1];
			d[j][0] = 0.5*(xo+xn);
			dt[j] = d[j][0];
			for (int i=1;i<nd-1;i++) {
				d[j][i] = xo+xn;
				xo = xn;
				xn = d[j][i+1];
				d[j][i] = (d[j][i]+xn)/3.;
				dt[j] += d[j][i];
			}
			d[j][nd-1] = 0.5*(xn+xo);
			dt[j] += d[j][nd-1];
		}
      
		for(int j=0;j<ndim;j++) {
			double rc = 0.;
			for(int i=0;i<nd;i++) {
				r[i] = 0.;
				if(d[j][i]>0.) {
					double xo = dt[j]/d[j][i];
					if(!isinf(xo)){
						r[i] = pow(((xo-1.)/xo/log(xo)),alph);
					}
				}
				rc += r[i];
			}
			rc /= xnd;
			int k = -1;
			double xn = 0.;
			double dr = xn;
			int i = k;
			k++;
			dr += r[k];
			double xo = xn;
			xn = xi[j][k];

			do{
				while (dr<=rc) {
					k++;
					dr += r[k];
					xo = xn;
					xn = xi[j][k];
				}
				i++;
				dr -= rc;
				xin[i] = xn-(xn-xo)*dr/r[k];
			} while (i<nd-2);

			for (int i=0;i<nd-1;i++) {
				xi[j][i] = (double)xin[i];
			}
			
			xi[j][nd-1] = 1.;
		}

		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
		hipDeviceSynchronize();

		timerVegasRefine.stop();
   } while (it<itmx && acc*fabs(avgi)<sd);

	gpuErrchk(hipHostFree(hFval));
	gpuErrchk(hipFree(gFval));

	gpuErrchk(hipHostFree(hIAval));
	gpuErrchk(hipFree(gIAval));

	avgi = 11.1;
	sd = 22.2;
	chi2a = 33.33;



	LOG_FUNC_END
}



}
}

