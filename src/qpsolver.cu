#include "hip/hip_runtime.h"
#include "qpsolver.h"

/* constructor */
QPSolver::QPSolver(Data* data, Gamma *gamma, Theta *theta, Scalar eps_sqr){
	this->data = data;
	this->gamma = gamma;
	this->theta = theta;
	this->eps_sqr = eps_sqr;	
}

/* prepare data which are constant */
void QPSolver::init(){
	int t,k;

	int T = this->get_T();
	int K = this->get_K();
	
	/* prepare block of hessian matrix */
	GammaMatrix<Scalar> A_sub(T,T); 
	for(t=0;t<T;t++){
		/* first row */
		if(t == 0){
			A_sub(t,t) = 1.0;
			A_sub(t,t+1) = -1.0;
		}
		/* common row */
		if(t > 0 && t < T-1){
			A_sub(t,t+1) = -1.0;
			A_sub(t,t) = 2.0;
			A_sub(t,t-1) = -1.0;
		}
		/* last row */
		if(t == T-1){
			A_sub(t,t-1) = -1.0;
			A_sub(t,t) = 1.0;
		}
	}	
	A_sub *= 0.5*this->eps_sqr;
	this->A_sub = A_sub;

	/* prepare RHS bs, gs, ds */
	this->bs = new GammaVector<Scalar>[K];
	this->gs = new GammaVector<Scalar>[K];
	this->ds = new GammaVector<Scalar>[K];
	this->Ads = new GammaVector<Scalar>[K];
	/* alloc first vector */
	DataVector<Scalar> b(T);
	/* set initial zero value to all vectors */
	b(all) = 0.0;
	for(k=0;k<K;k++){
		this->bs[k] = b;
		this->gs[k] = b;
		this->ds[k] = b;
		this->Ads[k] = b;
	}

}

void QPSolver::finalize(){
	/* clean the mess */
	delete []this->bs;
	delete []this->gs;
	delete []this->ds;
	delete []this->Ads;
}

void QPSolver::solve(){
	/* algorithm parameters */
	int m = ALGORITHM_SPGQP_m;
	Scalar gamma = ALGORITHM_SPGQP_gamma;
	Scalar sigma2 = ALGORITHM_SPGQP_sigma2;
	Scalar eps = ALGORITHM_SPGQP_eps;
	int maxit = ALGORITHM_SPGQP_maxit;
	Scalar alphainit = ALGORITHM_SPGQP_alphainit;

	/* output performance */
	int it = 0;
	int hess_mult = 0;

	int k; /* iterator through clusters */
	int K = this->get_K(); /* number of clusters */
	Scalar fx; /* function value */
	GammaVector<Scalar> fs(m); /* store function values for generalized A-condition */
	Scalar fx_max; /* max(fs) */
	Scalar xi, beta_bar, beta_hat,beta; /* for A-condition */
	Scalar dd; /* dot(d,d) */
	Scalar gd; /* dot(g,d) */
	Scalar dAd; /* dot(Ad,d) */
	Scalar alpha_bb; /* BB step-size */
	
	
	/* compute and set new RHS */
	/* b = -g(data,theta) */
	for(k=0;k<K;k++){ // TODO: parallel
		this->gamma->compute_gk(&(this->bs[k]), this->data, this->theta, k);
		this->bs[k] *= -1.0;
	}
	
	/* project initial approximation to feasible set */
	this->project(&(this->gamma->gamma_vecs));

	/* compute gradient, g = A*x-b */
	hess_mult += 1;
	for(k=0;k<K;k++){ // TODO: parallel
		this->gs[k] = this->A_sub*this->gamma->gamma_vecs[k]; 
		this->gs[k] -= this->bs[k];
	}

	/* compute function value */
	fx = get_function_value();
	fs(all) = fx;
	
	/* initial step-size */
	alpha_bb = alphainit;
	
	/* main cycle */
	while(it < maxit){
		/* d = x - alpha_bb*g, see next step, it will be d = P(x - alpha_bb*g) - x */
		/* Ad = A*d */
		/* dAd = dot(Ad,d) */
		/* gd = dot(g,d) */
		dAd = 0.0;
		gd = 0.0;
		for(k = 0; k < K;k++){ // TODO: parallel
			this->ds[k] = this->gamma->gamma_vecs[k] - alpha_bb*this->gs[k];
			this->Ads[k] = this->A_sub*this->ds[k];

			dAd += dot(this->Ads[k],this->ds[k]);
			gd += dot(this->gs[k],this->ds[k]);
		}
		
		/* d = P(d) */
		this->project(&(this->ds));
		
		/* d = d - x */
		/* dd = dot(d,d) */
		dd = 0.0;
		for(k = 0; k < K;k++){ // TODO: parallel
			this->ds[k] -= this->gamma->gamma_vecs[k];
			dd += dot(this->ds[k],this->ds[k]);
		}
		
		/* stopping criteria */
		if(dd < eps){
			break;
		}
		
		/* fx_max = max(fs) */
		fx_max = max(fs);	
		
		/* compute step-size from A-condition */
		xi = (fx_max - fx)/dAd;
		beta_bar = -gd/dAd;
		beta_hat = gamma*beta_bar + sqrt(gamma*gamma*beta_bar-beta_bar + 2*xi);

		/* beta = min(sigma2,beta_hat) */
		if(beta_hat < sigma2){
			beta = beta_hat;
		} else {
			beta = sigma2;
		}
		
		/* update approximation and gradient */
		/* x = x + beta*d */
		/* g = g + beta*Ad */
		for(k = 0; k < K;k++){ // TODO: parallel
			this->gamma->gamma_vecs[k] += beta*this->ds[k]; 
			this->gs[k] += beta*this->Ads[k];
		}
		
		/* compute new function value */
		fx = this->get_function_value();
		
		/* update fs */
		/* fs(1:end-1) = fs(2:end); */
		/* fs(end) = f;	*/
		if(m == 1){
			fs(0) = fx;
		} else {
			fs(0,m-2) = fs(1,m-1);
			fs(m-1) = fx;
		}
		
		/* update BB step-size */
		alpha_bb = dd/dAd;
		
		
		/* print progress of algorithm */
		if(DEBUG_ALGORITHM_PRINTFS || DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[33mit = \033[0m" << it << std::endl;
		}

		if(DEBUG_ALGORITHM_PRINTFS){
			std::cout << "\033[36m fs = \033[0m" << fs << std::endl;
		}
		
		if(DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[36m dd = \033[0m" << dd << ",";
			std::cout << "\033[36m dAd = \033[0m" << dAd << ",";
			std::cout << "\033[36m gd = \033[0m" << gd << std::endl;
			
			std::cout << "\033[36m fx = \033[0m" << fx << ",";
			std::cout << "\033[36m fx_max = \033[0m" << fx_max << ",";
			std::cout << "\033[36m xi = \033[0m" << xi << std::endl;
			
			std::cout << "\033[36m beta_bar = \033[0m" << beta_bar << ",";
			std::cout << "\033[36m beta_hat = \033[0m" << beta_hat << ",";
			std::cout << "\033[36m beta = \033[0m" << beta << std::endl;
			
			std::cout << "\033[36m alpha_bb = \033[0m" << alpha_bb << std::endl;
			
		}
		
		/* increase iteration counter */
		it += 1;
	}

}

void QPSolver::project(GammaVector<Scalar> **x){
	int t,k;
	GammaVector<Scalar> x_sub(this->get_K());

	for(t=0;t<this->get_T();t++){	// TODO: this is the place, where the parallel impementation should make a point
		/* cut x_sub from x */
		for(k=0;k<this->get_K();k++){
			x_sub(k) = (*x)[k](t);
		}
		
		/* compute subprojection */
		this->project_sub(&x_sub);

		/* add x_sub back to x */
		for(k=0;k<this->get_K();k++){
			(*x)[k](t) = x_sub(k);
		}
	}

}

/* project x_sub to feasible set defined by equality and inequality constraints
 * sum(x_sub) = 1
 * x_sub >= 0
 */
void QPSolver::project_sub(GammaVector<Scalar> *x_sub){
	int n = this->get_K(); /* nmb of components of x_sub */
	int i;

	bool is_inside = true;
	
	/* control equality constraints */
	if(sum(*x_sub) != 1){ 
		is_inside = false;
	}
	
	/* control inequality constraints */
	for(i = 0; i < n; i++){ // TODO: could be performed parallely  
		if((*x_sub)(i) < 0.0){
			is_inside = false;
		}
	}

	/* if given point is not inside the feasible domain, then do projection */
	if(!is_inside){
		/* compute sorted x_sub */
		GammaVector<Scalar> y(n);
		y = *x_sub; 
		this->sort_bubble(&y);

		/* now perform analytical solution of projection problem */	
		Scalar t_hat = 0.0;
		i = n - 1;
		Scalar ti;

		while(i >= 1){
			ti = (sum(y(i,n-1)) - 1.0)/(Scalar)(n-i);
			if(ti >= y(i-1)){
				t_hat = ti;
				break;
			} else {
				i = i - 1;
			}
		}

		if(i == 0){
			t_hat = (sum(y(0,n-1))-1.0)/(Scalar)n;
		}
    
		for(i = 0; i < n; i++){ // TODO: could be performed parallely  
			/* (*x_sub)(i) = max(*x_sub-t_hat,0); */
			ti = (*x_sub)(i) - t_hat;	
			if(ti > 0.0){
				(*x_sub)(i) = ti;
			} else {
				(*x_sub)(i) = 0.0;
			}
		}
	}
}

/* sort values of scalar vector */
void QPSolver::sort_bubble(GammaVector<Scalar> *x){
	int n = x->size();
	int i;
	int nnew;
	Scalar swap;

	while(n > 1){
		/* Iterate through x */
		nnew = 0;
		for(i=1;i<n;i++){
			/* Swap elements in wrong order */
			if ((*x)(i) < (*x)(i - 1)){
				swap = (*x)(i);
				(*x)(i) = (*x)(i-1);
				(*x)(i-1) = swap;
				nnew = i;
			}
        }
		n = nnew;
    }
}

Scalar QPSolver::get_function_value(){
	Scalar fx = 0.0;
	int k;
	for(k=0;k<this->get_K();k++){ // TODO: parallel
		 fx += 0.5*dot(this->gs[k]-this->bs[k],this->gamma->gamma_vecs[k]);
	}
	return fx;
}

void QPSolver::print(){
	this->print(0);
}

void QPSolver::print(int nmb_of_spaces){
	this->print(nmb_of_spaces, true);
}

void QPSolver::print(int nmb_of_spaces, bool print_A_sub){
	int i,k;
	int K = this->get_K();
	
	std::ostringstream oss_spaces;

	std::ostringstream oss;
	std::ostringstream oss_values;
	
	for(i=0;i<nmb_of_spaces;i++){
		oss_spaces << " ";
	}
	
	oss << oss_spaces.str() << "- QP optimization problem";
	Message_info(oss.str());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - K = ";
	Message_info_value(oss.str(),this->get_K());
	oss.str(""); oss.clear();
	
	oss << oss_spaces.str() << " - T = ";
	Message_info_value(oss.str(),this->get_T());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - dim = ";
	Message_info_value(oss.str(),this->get_dim());
	oss.str(""); oss.clear();

	if(print_A_sub){
		oss << oss_spaces.str() << " - block of Hessian matrix Asub:";
		Message_info(oss.str());
		oss.str(""); oss.clear();
		oss_values << this->A_sub;
		Message_info_values(oss.str(),oss_values.str());	
		oss_values.str(""); oss_values.clear();
	}

	oss << oss_spaces.str() << " - right hand-side vector b:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   b[" << k << "] = ";
		oss_values << this->bs[k];
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

	oss << oss_spaces.str() << " - vector of unknowns x:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   x[" << k << "] = ";
		oss_values << this->gamma->gamma_vecs[k];
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

}

int QPSolver::get_T(){
	return this->data->get_T();
}

int QPSolver::get_dim(){
	return this->data->get_dim();
}

int QPSolver::get_K(){
	return this->gamma->get_K();
}


