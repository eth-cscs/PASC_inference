#include "hip/hip_runtime.h"
#include "qpsolver.h"

/* SPGQP SETTINGS */
#define ALGORITHM_SPGQP_m 1
#define ALGORITHM_SPGQP_gamma 0.9
#define ALGORITHM_SPGQP_sigma2 1.0
#define ALGORITHM_SPGQP_eps 0.0001
#define ALGORITHM_SPGQP_maxit 10000
#define ALGORITHM_SPGQP_lambdaest 4.0
#define DEBUG_ALGORITHM_BASIC true /* basic information about used algorithm and parameters */
#define DEBUG_ALGORITHM_SAYHELLO false /* information about used algorithm and parameters */
#define DEBUG_ALGORITHM_PRINTF false /* print object function value in every iteration */
#define DEBUG_ALGORITHM_PRINTFS false /* print vector of object functions in every iteration */
#define DEBUG_ALGORITHM_PRINTCOEFF true /* print computed coefficients in every iteration */


/* constructor */
QPSolver::QPSolver(Data* data, Gamma *gamma, Theta *theta, Scalar eps_sqr){
	this->data = data;
	this->gamma = gamma;
	this->theta = theta;
	this->eps_sqr = eps_sqr;	
}

/* prepare data which are constant */
void QPSolver::init(){
	int t,k;

	int T = this->get_T();
	int K = this->get_K();
	
	/* prepare block of hessian matrix */
	GammaMatrix<Scalar> A_sub(T,T); 
	for(t=0;t<T;t++){
		/* first row */
		if(t == 0){
			A_sub(t,t) = 1.0;
			A_sub(t,t+1) = -1.0;
		}
		/* common row */
		if(t > 0 && t < T-1){
			A_sub(t,t+1) = -1.0;
			A_sub(t,t) = 2.0;
			A_sub(t,t-1) = -1.0;
		}
		/* last row */
		if(t == T-1){
			A_sub(t,t-1) = -1.0;
			A_sub(t,t) = 1.0;
		}
	}	
	A_sub *= this->eps_sqr;
	this->A_sub = A_sub;

	/* prepare RHS bs, gs, ds */
	this->bs = new GammaVector<Scalar>[K];
	this->gs = new GammaVector<Scalar>[K];
	this->ds = new GammaVector<Scalar>[K];
	this->Ads = new GammaVector<Scalar>[K];
	/* alloc first vector */
	DataVector<Scalar> b(T);
	/* set initial zero value to all vectors */
	b(all) = 0.0;
	for(k=0;k<K;k++){
		this->bs[k] = b;
		this->gs[k] = b;
		this->ds[k] = b;
		this->Ads[k] = b;
	}

}

void QPSolver::finalize(){
	/* clean the mess */
	delete []this->bs;
	delete []this->gs;
	delete []this->ds;
	delete []this->Ads;
}

void QPSolver::compute_b(){
	int k;
	for(k=0;k<this->gamma->get_K();k++){ // TODO: parallel
		this->gamma->compute_gk(&(this->bs[k]), this->data, this->theta, k);
		this->bs[k] *= -1.0;
	}
}

void QPSolver::solve(){
	/* algorithm parameters */
	int m = ALGORITHM_SPGQP_m;
	Scalar gamma = ALGORITHM_SPGQP_gamma;
	Scalar sigma2 = ALGORITHM_SPGQP_sigma2;
	Scalar eps = ALGORITHM_SPGQP_eps;
	int maxit = ALGORITHM_SPGQP_maxit;
	Scalar alphainit = 1.0/(this->eps_sqr*ALGORITHM_SPGQP_lambdaest);

	/* output performance */
	int it = 0;
	int hess_mult = 0;
	Scalar comp_time;
	timer.start();

	int k; /* iterator through clusters */
	int K = this->get_K(); /* number of clusters */
	Scalar fx; /* function value */
	GammaVector<Scalar> fs(m); /* store function values for generalized A-condition */
	Scalar fx_max; /* max(fs) */
	Scalar xi, beta_bar, beta_hat,beta; /* for A-condition */
	Scalar dd; /* dot(d,d) */
	Scalar gd; /* dot(g,d) */
	Scalar dAd; /* dot(Ad,d) */
	Scalar alpha_bb; /* BB step-size */
	
	// TODO: temp
	Scalar fx_orig;
	Scalar fx_final;
	
	
	/* compute and set new RHS */
	/* b = -g(data,theta) */
	this->compute_b();

	fx_orig = this->get_function_value();
	
	/* project initial approximation to feasible set */
//	this->project(&(this->gamma->gamma_vecs));

	/* compute gradient, g = A*x-b */
	hess_mult += 1;
	for(k=0;k<K;k++){ // TODO: parallel
		this->gs[k] = this->A_sub*this->gamma->gamma_vecs[k]; 
		this->gs[k] -= this->bs[k];
	}
	
//	std::cout << this->A_sub << std::endl;
	
//	std::cout << "g:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->gs[k] << std::endl;
//	}

	/* compute function value */
	fx = this->get_function_value();
	fs(all) = fx;

//	std::cout << "fs:" << fs << std::endl;
	
	/* initial step-size */
	alpha_bb = alphainit;

//	std::cout << "alpha_bb:" << alphainit << std::endl;

	/* print basic informations about algorithm */
	if(DEBUG_ALGORITHM_SAYHELLO){
		Message_info("- SPGQP BEGIN -------------------------------------------------------------");
		Message_info_main("- parameters:");
		Message_info_value(" - m = \t\t\t",m);
		Message_info_value(" - gamma = \t\t",gamma);
		Message_info_value(" - sigma2 = \t\t",sigma2);
		Message_info_value(" - alpha_init = \t",alphainit);
		Message_info_value(" - init fx = \t\t",fx);
		Message_info_value(" - eps = \t\t",eps);
		Message_info_value(" - maxit = \t\t",maxit);
	}
	
	/* main cycle */
	while(it < maxit){
		/* d = x - alpha_bb*g, see next step, it will be d = P(x - alpha_bb*g) - x */
		for(k = 0; k < K;k++){ // TODO: parallel
			this->ds[k] = this->gamma->gamma_vecs[k] - alpha_bb*this->gs[k];
		}

//	std::cout << "d1:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->ds[k] << std::endl;
//	}
		
		/* d = P(d) */
		this->project(&(this->ds));

//	std::cout << "d2:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->ds[k] << std::endl;
//	}
		
		/* d = d - x */
		/* Ad = A*d */
		/* dd = dot(d,d) */
		/* dAd = dot(Ad,d) */
		/* gd = dot(g,d) */
		dd = 0.0;
		dAd = 0.0;
		gd = 0.0;
		hess_mult+=1;
		for(k = 0; k < K;k++){ // TODO: parallel
			this->ds[k] -= this->gamma->gamma_vecs[k];
			this->Ads[k] = this->A_sub*this->ds[k];

			dd += dot(this->ds[k],this->ds[k]);
			dAd += dot(this->Ads[k],this->ds[k]);
			gd += dot(this->gs[k],this->ds[k]);
		}

//	std::cout << "d:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->ds[k] << std::endl;
//	}
//	std::cout << "Ad:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->Ads[k] << std::endl;
//	}
//	std::cout << "dd:" << dd << std::endl;
//	std::cout << "dAd:" << dAd << std::endl;
//	std::cout << "gd:" << gd << std::endl;
		
		/* stopping criteria */
		if(dd < eps){
			break;
		}
		
		/* fx_max = max(fs) */
		fx_max = max(fs);	
		
		/* compute step-size from A-condition */
		xi = (fx_max - fx)/dAd;
		beta_bar = -gd/dAd;
		beta_hat = beta_bar; //gamma*beta_bar + sqrt(gamma*gamma*beta_bar*beta_bar + 2*xi);

//	std::cout << "fmax:" << fx_max << std::endl;
//	std::cout << "xi:" << xi << std::endl;
//	std::cout << "beta_bar:" << beta_bar << std::endl;
//	std::cout << "beta_hat:" << beta_hat << std::endl;

		/* beta = min(sigma2,beta_hat) */
		if(beta_hat < sigma2){
			beta = beta_hat;
		} else {
			beta = sigma2;
		}

//	std::cout << "beta:" << beta << std::endl;
		
		/* update approximation and gradient */
		/* x = x + beta*d */
		/* g = g + beta*Ad */

		std::cout << "fx_before:" << this->get_function_value() << std::endl;
		for(k = 0; k < K;k++){ // TODO: parallel
			this->gamma->gamma_vecs[k] += beta*this->ds[k]; 
//			this->gs[k] += beta*this->Ads[k];

			this->gs[k] = this->A_sub*this->gamma->gamma_vecs[k];
			this->gs[k] += -this->bs[k];

		}
		std::cout << "fx_after:" << this->get_function_value() << std::endl;

		

//	std::cout << "x:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->gamma->gamma_vecs[k] << std::endl;
//	}

//	std::cout << "g:" << std::endl;
//	for(k=0;k<K;k++){ // TODO: parallel
//		std::cout << this->gs[k] << std::endl;
//	}
		
		/* compute new function value */
		fx = this->get_function_value();

		
		/* update fs */
		/* fs(1:end-1) = fs(2:end); */
		/* fs(end) = f;	*/
		if(m == 1){
			fs(0) = fx;
		} else {
			fs(0,m-2) = fs(1,m-1);
			fs(m-1) = fx;
		}

//	std::cout << "fs:" << fs << std::endl;
		
		/* update BB step-size */
		alpha_bb = dd/dAd;

//	std::cout << "alpha_bb:" << alpha_bb << std::endl;
		
		/* print progress of algorithm */
		if(DEBUG_ALGORITHM_PRINTF || DEBUG_ALGORITHM_PRINTFS || DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[33mit = \033[0m" << it << std::endl;
		}

		if(DEBUG_ALGORITHM_PRINTF){
			std::cout << "\033[36m fx = \033[0m" << fx << std::endl;
		}

		if(DEBUG_ALGORITHM_PRINTFS){
			std::cout << "\033[36m fs = \033[0m" << fs << std::endl;
		}
		
		if(DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[36m dd = \033[0m" << dd << ",";
			std::cout << "\033[36m dAd = \033[0m" << dAd << ",";
			std::cout << "\033[36m gd = \033[0m" << gd << std::endl;
			
			std::cout << "\033[36m fx = \033[0m" << fx << ",";
			std::cout << "\033[36m fx_max = \033[0m" << fx_max << ",";
			std::cout << "\033[36m xi = \033[0m" << xi << std::endl;
			
			std::cout << "\033[36m beta_bar = \033[0m" << beta_bar << ",";
			std::cout << "\033[36m beta_hat = \033[0m" << beta_hat << ",";
			std::cout << "\033[36m beta = \033[0m" << beta << std::endl;
			
			std::cout << "\033[36m alpha_bb = \033[0m" << alpha_bb << std::endl;
			
		}
		
		/* increase iteration counter */
		it += 1;
	} /* main cycle end */

	comp_time = timer.stop();
	/* say goodbye */
	if(DEBUG_ALGORITHM_SAYHELLO){
		Message_info_main("\n- final info:");
		Message_info_time(" - time: \t\t",comp_time);
		Message_info_value(" - it: \t\t\t",it);
		Message_info_value(" - hess_mult: \t\t",hess_mult);
		Message_info_value(" - final fx = \t\t",fx);
		Message_info("- SPGQP END ---------------------------------------------------------------");
	}

	fx_final = this->get_function_value();

	/* very short info */
	if(DEBUG_ALGORITHM_BASIC){
		Message_info("  - SPGQP algorithm");
		Message_info_value("   - it    = ",it);
		Message_info_time("   - time  = ",comp_time);

		Message_info_value("   - f_orig   = ",fx_orig);
		Message_info_value("   - f_final  = ",fx_final);

	}

}

void QPSolver::project(GammaVector<Scalar> **x){
	int t,k;
	GammaVector<Scalar> x_sub(this->get_K());

	for(t=0;t<this->get_T();t++){	// TODO: this is the place, where the parallel impementation should make a point
		/* cut x_sub from x */
		for(k=0;k<this->get_K();k++){
			x_sub(k) = (*x)[k](t);
		}
		
		/* compute subprojection */
		this->project_sub(&x_sub);

		/* add x_sub back to x */
		for(k=0;k<this->get_K();k++){
			(*x)[k](t) = x_sub(k);
		}
	}

}

/* project x_sub to feasible set defined by equality and inequality constraints
 * sum(x_sub) = 1
 * x_sub >= 0
 */
void QPSolver::project_sub(GammaVector<Scalar> *x_sub){
	int n = this->get_K(); /* nmb of components of x_sub */
	int i;

	bool is_inside = true;
	
	/* control equality constraints */
	if(sum(*x_sub) != 1){ 
		is_inside = false;
	}
	
	/* control inequality constraints */
	for(i = 0; i < n; i++){ // TODO: could be performed parallely  
		if((*x_sub)(i) < 0.0){
			is_inside = false;
		}
	}

	/* if given point is not inside the feasible domain, then do projection */
	if(!is_inside){
		/* compute sorted x_sub */
		GammaVector<Scalar> y(n);
		for(i=0;i<n;i++){ // TODO: it is really necessary?
			y(i) = (*x_sub)(i); 
		}
		this->sort_bubble(&y);

		/* now perform analytical solution of projection problem */	
		Scalar t_hat = 0.0;
		i = n - 1;
		Scalar ti;

		while(i >= 1){
			ti = (sum(y(i,n-1)) - 1.0)/(Scalar)(n-i);
			if(ti >= y(i-1)){
				t_hat = ti;
				i = -1; /* break */
			} else {
				i = i - 1;
			}
		}

		if(i == 0){
			t_hat = (sum(y)-1.0)/(Scalar)n;
		}
    
		for(i = 0; i < n; i++){ // TODO: could be performed parallely  
			/* (*x_sub)(i) = max(*x_sub-t_hat,0); */
			ti = (*x_sub)(i) - t_hat;	
			if(ti > 0.0){
				(*x_sub)(i) = ti;
			} else {
				(*x_sub)(i) = 0.0;
			}
		}
	}
}

/* sort values of scalar vector */
void QPSolver::sort_bubble(GammaVector<Scalar> *x){
	int n = x->size();
	int i;
	int nnew;
	Scalar swap;

	while(n > 0){
		/* Iterate through x */
		nnew = 0;
		for(i=1;i<n;i++){
			/* Swap elements in wrong order */
			if ((*x)(i) < (*x)(i - 1)){
				swap = (*x)(i);
				(*x)(i) = (*x)(i-1);
				(*x)(i-1) = swap;
				nnew = i;
			}
        }
		n = nnew;
    }
}

Scalar QPSolver::get_function_value(){
	Scalar fx = 0.0;
	int k;
/*	for(k=0;k<this->get_K();k++){ // TODO: parallel
		 fx += 0.5*dot(this->gs[k]-this->bs[k],this->gamma->gamma_vecs[k]);
	}
*/ 

	GammaVector<Scalar> temp(this->get_T());

	for(k=0;k<this->get_K();k++){ // TODO: parallel
		 temp = this->A_sub*this->gamma->gamma_vecs[k];
		 fx += 0.5*dot(temp,this->gamma->gamma_vecs[k]);
		 fx -= dot(this->gamma->gamma_vecs[k],this->bs[k]);
	}	

	return fx;
}

void QPSolver::print(){
	this->print(0);
}

void QPSolver::print(int nmb_of_spaces){
	this->print(nmb_of_spaces, true);
}

void QPSolver::print(int nmb_of_spaces, bool print_A_sub){
	int i,k;
	int K = this->get_K();
	
	std::ostringstream oss_spaces;

	std::ostringstream oss;
	std::ostringstream oss_values;
	
	for(i=0;i<nmb_of_spaces;i++){
		oss_spaces << " ";
	}
	
	oss << oss_spaces.str() << "- QP optimization problem";
	Message_info(oss.str());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - K = ";
	Message_info_value(oss.str(),this->get_K());
	oss.str(""); oss.clear();
	
	oss << oss_spaces.str() << " - T = ";
	Message_info_value(oss.str(),this->get_T());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - dim = ";
	Message_info_value(oss.str(),this->get_dim());
	oss.str(""); oss.clear();

	if(print_A_sub){
		oss << oss_spaces.str() << " - block of Hessian matrix Asub:";
		Message_info(oss.str());
		oss.str(""); oss.clear();
		oss_values << this->A_sub;
		Message_info_values(oss.str(),oss_values.str());	
		oss_values.str(""); oss_values.clear();
	}

	oss << oss_spaces.str() << " - right hand-side vector b:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   b[" << k << "] = ";
		oss_values << this->bs[k];
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

	oss << oss_spaces.str() << " - vector of unknowns x:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   x[" << k << "] = ";
		oss_values << this->gamma->gamma_vecs[k];
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

}

int QPSolver::get_T(){
	return this->data->get_T();
}

int QPSolver::get_dim(){
	return this->data->get_dim();
}

int QPSolver::get_K(){
	return this->gamma->get_K();
}


