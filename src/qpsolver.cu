#include "hip/hip_runtime.h"
#include "qpsolver.h"

/* SPGQP SETTINGS */
#define ALGORITHM_SPGQP_m 30
#define ALGORITHM_SPGQP_gamma 0.9
#define ALGORITHM_SPGQP_sigma2 1.0
#define ALGORITHM_SPGQP_eps 0.0001
#define ALGORITHM_SPGQP_maxit 10000
#define ALGORITHM_SPGQP_lambdaest 4.0
#define DEBUG_ALGORITHM_BASIC false /* basic information about used algorithm and parameters */
#define DEBUG_ALGORITHM_SAYHELLO false /* information about used algorithm and parameters */
#define DEBUG_ALGORITHM_PRINTF false /* print object function value in every iteration */
#define DEBUG_ALGORITHM_PRINTFS false /* print vector of object functions in every iteration */
#define DEBUG_ALGORITHM_PRINTCOEFF false /* print computed coefficients in every iteration */


/* constructor */
QPSolver::QPSolver(Data* data, Gamma *gamma, Theta *theta, Scalar eps_sqr){
	this->data = data;
	this->gamma = gamma;
	this->theta = theta;
	this->eps_sqr = eps_sqr;	
}

/* prepare data which are constant */
void QPSolver::init(){
	int k;

	int T = this->get_T();
	int K = this->get_K();
	
	/* prepare RHS bs, gs, ds */
	this->bs = new GammaVector<Scalar>[K];
	this->gs = new GammaVector<Scalar>[K];
	this->ds = new GammaVector<Scalar>[K];
	this->Ads = new GammaVector<Scalar>[K];
	/* alloc first vector */
	DataVector<Scalar> b(T);
	/* set initial zero value to all vectors */
	b(all) = 0.0;
	for(k=0;k<K;k++){
		this->bs[k] = b;
		this->gs[k] = b;
		this->ds[k] = b;
		this->Ads[k] = b;
	}

}

void QPSolver::finalize(){
	/* clean the mess */
	delete []this->bs;
	delete []this->gs;
	delete []this->ds;
	delete []this->Ads;
}

void QPSolver::get_Ax(GammaVector<double> *Ax, GammaVector<double> x){
	int N = x.size();
	int t;

	for(t=0;t<N;t++){
		/* first row */
		if(t == 0){
			(*Ax)(t) = x(t) - x(t+1);
		}
		/* common row */
		if(t > 0 && t < N-1){
			(*Ax)(t) = -x(t-1) + 2.0*x(t) - x(t+1);
		}
		/* last row */
		if(t == N-1){
			(*Ax)(t) = -x(t-1) + x(t);
		}
	}
}

void QPSolver::compute_b(){
	int k;
	for(k=0;k<this->gamma->get_K();k++){ // TODO: parallel
		this->gamma->compute_gk(&(this->bs[k]), this->data, this->theta, k);
		this->bs[k] *= -1.0;
	}
}

void QPSolver::solve(){
	/* algorithm parameters */
	int m = ALGORITHM_SPGQP_m;
	Scalar gamma = ALGORITHM_SPGQP_gamma;
	Scalar sigma2 = ALGORITHM_SPGQP_sigma2;
	Scalar eps = ALGORITHM_SPGQP_eps;
	int maxit = ALGORITHM_SPGQP_maxit;
	Scalar alphainit = 1.0/(this->eps_sqr*ALGORITHM_SPGQP_lambdaest);

	/* output performance */
	this->it = 0;
	int hess_mult = 0;
	Scalar comp_time;
	timer.start();

	int k; /* iterator through clusters */
	int K = this->get_K(); /* number of clusters */
	Scalar fx; /* function value */
	GammaVector<Scalar> fs(m); /* store function values for generalized A-condition */
	Scalar fx_max; /* max(fs) */
	Scalar xi, beta_bar, beta_hat,beta; /* for A-condition */
	Scalar dd; /* dot(d,d) */
	Scalar gd; /* dot(g,d) */
	Scalar dAd; /* dot(Ad,d) */
	Scalar alpha_bb; /* BB step-size */
	
	/* compute and set new RHS */
	/* b = -g(data,theta) */
	this->compute_b();

	/* project initial approximation to feasible set */
	this->project(&(this->gamma->gamma_vecs));

	/* compute gradient, g = A*x-b */
	hess_mult += 1;
	for(k=0;k<K;k++){ // TODO: parallel
		get_Ax(&(this->gs[k]),this->gamma->gamma_vecs[k]); 
		this->gs[k] -= this->bs[k];
	}
	
	/* compute function value */
	fx = this->get_function_value(this->gamma->gamma_vecs);
	fs(all) = fx;
	
	/* initial step-size */
	alpha_bb = alphainit;

	/* print basic informations about algorithm */
	if(DEBUG_ALGORITHM_SAYHELLO){
		Message_info("- SPGQP BEGIN -------------------------------------------------------------");
		Message_info_main("- parameters:");
		Message_info_value(" - m = \t\t\t",m);
		Message_info_value(" - gamma = \t\t",gamma);
		Message_info_value(" - sigma2 = \t\t",sigma2);
		Message_info_value(" - alpha_init = \t",alphainit);
		Message_info_value(" - init fx = \t\t",fx);
		Message_info_value(" - eps = \t\t",eps);
		Message_info_value(" - maxit = \t\t",maxit);
	}
	
	/* main cycle */
	while(this->it < maxit){
		/* d = x - alpha_bb*g, see next step, it will be d = P(x - alpha_bb*g) - x */
		for(k = 0; k < K;k++){ // TODO: parallel
			this->ds[k] = this->gamma->gamma_vecs[k] - alpha_bb*(this->gs[k]);
		}

		/* d = P(d) */
		this->project(&(this->ds));
		
		/* d = d - x */
		/* Ad = A*d */
		/* dd = dot(d,d) */
		/* dAd = dot(Ad,d) */
		/* gd = dot(g,d) */
		dd = 0.0;
		dAd = 0.0;
		gd = 0.0;
		hess_mult+=1;
		for(k = 0; k < K;k++){ // TODO: parallel
			this->ds[k] += -this->gamma->gamma_vecs[k];
			get_Ax(&(this->Ads[k]),this->ds[k]);

			dd += dot(this->ds[k],this->ds[k]);
			dAd += dot(this->Ads[k],this->ds[k]);
			gd += dot(this->gs[k],this->ds[k]);
		}
		
		/* stopping criteria */
		if(dd < eps){
			break;
		}
		
		/* fx_max = max(fs) */
		fx_max = max(fs);	
		
		/* compute step-size from A-condition */
		xi = (fx_max - fx)/dAd;
		beta_bar = -gd/dAd;
		beta_hat = gamma*beta_bar + sqrt(gamma*gamma*beta_bar*beta_bar + 2*xi);

		/* beta = min(sigma2,beta_hat) */
		if(beta_hat < sigma2){
			beta = beta_hat;
		} else {
			beta = sigma2;
		}

		/* update approximation and gradient */
		/* x = x + beta*d */
		/* g = g + beta*Ad */
		for(k = 0; k < K;k++){ // TODO: parallel
			this->gamma->gamma_vecs[k] += (this->ds[k])*beta; 

			/* use recursive formula to compute gradient */
			this->gs[k] += (this->Ads[k])*beta;

//			this->gs[k] = this->A_sub*this->gamma->gamma_vecs[k];
//			this->gs[k] -= this->bs[k];

		}
		
		/* compute new function value using gradient */
		fx = this->get_function_value(this->gamma->gamma_vecs, true);
		
		/* update fs */
		/* fs(1:end-1) = fs(2:end); */
		/* fs(end) = f;	*/
		if(m == 1){
			fs(0) = fx;
		} else {
			fs(0,m-2) = fs(1,m-1);
			fs(m-1) = fx;
		}
		
		/* update BB step-size */
		alpha_bb = dd/dAd;
		
		/* print progress of algorithm */
		if(DEBUG_ALGORITHM_PRINTF || DEBUG_ALGORITHM_PRINTFS || DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[33mit = \033[0m" << this->it << std::endl;
		}

		if(DEBUG_ALGORITHM_PRINTF){
			std::cout << "\033[36m fx = \033[0m" << fx << std::endl;
		}

		if(DEBUG_ALGORITHM_PRINTFS){
			std::cout << "\033[36m fs = \033[0m" << fs << std::endl;
		}
		
		if(DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[36m dd = \033[0m" << dd << ",";
			std::cout << "\033[36m dAd = \033[0m" << dAd << ",";
			std::cout << "\033[36m gd = \033[0m" << gd << std::endl;
			
			std::cout << "\033[36m fx = \033[0m" << fx << ",";
			std::cout << "\033[36m fx_max = \033[0m" << fx_max << ",";
			std::cout << "\033[36m xi = \033[0m" << xi << std::endl;
			
			std::cout << "\033[36m beta_bar = \033[0m" << beta_bar << ",";
			std::cout << "\033[36m beta_hat = \033[0m" << beta_hat << ",";
			std::cout << "\033[36m beta = \033[0m" << beta << std::endl;
			
			std::cout << "\033[36m alpha_bb = \033[0m" << alpha_bb << std::endl;
			
		}
		
		/* increase iteration counter */
		this->it += 1;
	} /* main cycle end */

	comp_time = timer.stop();
	/* say goodbye */
	if(DEBUG_ALGORITHM_SAYHELLO){
		Message_info_main("\n- final info:");
		Message_info_time(" - time: \t\t",comp_time);
		Message_info_value(" - it: \t\t\t",this->it);
		Message_info_value(" - hess_mult: \t\t",hess_mult);
		Message_info_value(" - final fx = \t\t",fx);
		Message_info("- SPGQP END ---------------------------------------------------------------");
	}

	/* very short info */
	if(DEBUG_ALGORITHM_BASIC){
		Message_info("  - SPGQP algorithm");
		Message_info_value("   - it    = ",this->it);
		Message_info_time("   - time  = ",comp_time);

	}

}

void QPSolver::project(GammaVector<Scalar> **x){
	int t,k;
	GammaVector<Scalar> x_sub(this->get_K());

	for(t=0;t<this->get_T();t++){	// TODO: this is the place, where the parallel impementation should make a point
		/* cut x_sub from x */
		for(k=0;k<this->get_K();k++){
			x_sub(k) = (*x)[k](t);
		}
		
		/* compute subprojection */
		this->project_sub(&x_sub);

		/* add x_sub back to x */
		for(k=0;k<this->get_K();k++){
			(*x)[k](t) = x_sub(k);
		}
	}

}

/* project x_sub to feasible set defined by equality and inequality constraints
 * sum(x_sub) = 1
 * x_sub >= 0
 */
void QPSolver::project_sub(GammaVector<Scalar> *x_sub){
	int n = this->get_K(); /* nmb of components of x_sub */
	int i;

	bool is_inside = true;
	
	/* control equality constraints */
	if(sum(*x_sub) != 1){ 
		is_inside = false;
	}
	
	/* control inequality constraints */
	for(i = 0; i < n; i++){ // TODO: could be performed parallely  
		if((*x_sub)(i) < 0.0){
			is_inside = false;
		}
	}

	/* if given point is not inside the feasible domain, then do projection */
	if(!is_inside){
		/* compute sorted x_sub */
		GammaVector<Scalar> y(n);
		for(i=0;i<n;i++){ // TODO: it is really necessary?
			y(i) = (*x_sub)(i); 
		}
		this->sort_bubble(&y);

		/* now perform analytical solution of projection problem */	
		Scalar t_hat = 0.0;
		i = n - 1;
		Scalar ti;

		while(i >= 1){
			ti = (sum(y(i,n-1)) - 1.0)/(Scalar)(n-i);
			if(ti >= y(i-1)){
				t_hat = ti;
				i = -1; /* break */
			} else {
				i = i - 1;
			}
		}

		if(i == 0){
			t_hat = (sum(y)-1.0)/(Scalar)n;
		}
    
		for(i = 0; i < n; i++){ // TODO: could be performed parallely  
			/* (*x_sub)(i) = max(*x_sub-t_hat,0); */
			ti = (*x_sub)(i) - t_hat;	
			if(ti > 0.0){
				(*x_sub)(i) = ti;
			} else {
				(*x_sub)(i) = 0.0;
			}
		}
	}
}

/* sort values of scalar vector */
void QPSolver::sort_bubble(GammaVector<Scalar> *x){
	int n = x->size();
	int i;
	int nnew;
	Scalar swap;

	while(n > 0){
		/* Iterate through x */
		nnew = 0;
		for(i=1;i<n;i++){
			/* Swap elements in wrong order */
			if ((*x)(i) < (*x)(i - 1)){
				swap = (*x)(i);
				(*x)(i) = (*x)(i-1);
				(*x)(i-1) = swap;
				nnew = i;
			}
        }
		n = nnew;
    }
}

Scalar QPSolver::get_function_value(GammaVector<Scalar> *x){
	return this->get_function_value(x,false);
}

Scalar QPSolver::get_function_value(GammaVector<Scalar> *x, bool use_gradient){
	Scalar fx;
	int k;

	if(use_gradient){
		/* use computed gradient in this->gs to compute function value */
		for(k=0;k<this->get_K();k++){ // TODO: parallel
			fx += 0.5*dot(this->gs[k]-this->bs[k],this->gamma->gamma_vecs[k]);
		}
	} else {
		/* we have nothing - compute fx using full formula fx = 0.5*dot(A*x,x) - dot(b,x) */
		
		GammaVector<Scalar> Ax(this->get_T());
		Scalar xAx, xb;

		fx = 0.0;
		for(k=0;k<this->get_K();k++){ // TODO: parallel
			get_Ax(&Ax,x[k]);
		 
			xAx = dot(Ax,x[k]);
			fx += 0.5*xAx;
		 
			xb = dot(x[k],this->bs[k]);
			fx -= xb;
		}

	}	

	return fx;	
}


void QPSolver::print(){
	this->print(0);
}

void QPSolver::print(int nmb_of_spaces){
	int i,k;
	int K = this->get_K();
	
	std::ostringstream oss_spaces;

	std::ostringstream oss;
	std::ostringstream oss_values;
	
	for(i=0;i<nmb_of_spaces;i++){
		oss_spaces << " ";
	}
	
	oss << oss_spaces.str() << "- QP optimization problem";
	Message_info(oss.str());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - K = ";
	Message_info_value(oss.str(),this->get_K());
	oss.str(""); oss.clear();
	
	oss << oss_spaces.str() << " - T = ";
	Message_info_value(oss.str(),this->get_T());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - dim = ";
	Message_info_value(oss.str(),this->get_dim());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - right hand-side vector b:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   b[" << k << "] = ";
		oss_values << this->bs[k];
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

	oss << oss_spaces.str() << " - vector of unknowns x:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   x[" << k << "] = ";
		oss_values << this->gamma->gamma_vecs[k];
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

}

int QPSolver::get_T(){
	return this->data->get_T();
}

int QPSolver::get_dim(){
	return this->data->get_dim();
}

int QPSolver::get_K(){
	return this->gamma->get_K();
}

int QPSolver::get_it(){
	return this->it;
}

