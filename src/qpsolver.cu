#include "hip/hip_runtime.h"
#include "qpsolver.h"

/* SPGQP SETTINGS */
#define ALGORITHM_SPGQP_m 30
#define ALGORITHM_SPGQP_gamma 0.9
#define ALGORITHM_SPGQP_sigma2 1.0
#define ALGORITHM_SPGQP_eps 0.00001
#define ALGORITHM_SPGQP_maxit 10000
#define ALGORITHM_SPGQP_lambdaest 4.0
#define DEBUG_ALGORITHM_BASIC false /* basic information about used algorithm and parameters */
#define DEBUG_ALGORITHM_SAYHELLO false /* information about used algorithm and parameters */
#define DEBUG_ALGORITHM_PRINTF false /* print object function value in every iteration */
#define DEBUG_ALGORITHM_PRINTFS false /* print vector of object functions in every iteration */
#define DEBUG_ALGORITHM_PRINTCOEFF false /* print computed coefficients in every iteration */

/* prepare data which are constant */
void QPSolver::init(int T, int K, Scalar eps_sqr){
	this->T = T;
	this->K = K;
	this->eps_sqr = eps_sqr;

	this->timer_projection.restart();
	this->timer_matmult.restart();
	this->timer_dot.restart();
	this->timer_update.restart();
	this->timer_stepsize.restart();
	this->timer_fs.restart();
	this->timer_total.restart();
	

	/* the time for initialization is the part of total time, it is necessary to add it */
	this->timer_total.start(); 
	
	/* prepare RHS bs, gs, ds */

	/* alloc first vector */
	DataVector b(this->K*this->T);
	/* set initial zero value to all vectors */
	b(all) = 0.0;

	this->b = b;
	this->g = b;
	this->d = b;
	this->Ad = b;

	this->it_all = 0;
	this->hessmult_all = 0;

	this->timer_total.stop();
}

void QPSolver::finalize(){
	/* the time is the part of total time, it is necessary to add it */
	this->timer_total.start(); 

	/* clean the mess */

	this->timer_total.stop();
}

void QPSolver::solve(GammaVector &x){
	this->timer_total.start(); /* stop this timer in the end of solution */

	/* algorithm parameters */
	int m = ALGORITHM_SPGQP_m;
	Scalar gamma = ALGORITHM_SPGQP_gamma;
	Scalar sigma2 = ALGORITHM_SPGQP_sigma2;
	Scalar eps = ALGORITHM_SPGQP_eps;
	int maxit = ALGORITHM_SPGQP_maxit;
	Scalar alphainit = 1.0/(this->eps_sqr*ALGORITHM_SPGQP_lambdaest);

	/* output performance */
	this->it = 0;
	this->hessmult = 0;

	int K = this->get_K(); /* number of clusters */
	Scalar fx; /* function value */
	GammaVector fs(m); /* store function values for generalized A-condition */
	Scalar fx_max; /* max(fs) */
	Scalar xi, beta_bar, beta_hat,beta; /* for A-condition */
	Scalar dd; /* dot(d,d) */
	Scalar gd; /* dot(g,d) */
	Scalar dAd; /* dot(Ad,d) */
	Scalar alpha_bb; /* BB step-size */

	
	/* initial step-size */
	alpha_bb = alphainit;

	/* print basic informations about algorithm */
	if(DEBUG_ALGORITHM_SAYHELLO){
		Message_info("- SPGQP BEGIN -------------------------------------------------------------");
		Message_info_main("- parameters:");
		Message_info_value(" - m = \t\t\t",m);
		Message_info_value(" - gamma = \t\t",gamma);
		Message_info_value(" - sigma2 = \t\t",sigma2);
		Message_info_value(" - alpha_init = \t",alphainit);
		Message_info_value(" - eps = \t\t",eps);
		Message_info_value(" - maxit = \t\t",maxit);
	}

	/* project initial approximation to feasible set */
	this->timer_projection.start();
	 get_projection(x, this->get_K());
	this->timer_projection.stop();

	/* compute gradient, g = A*x-b */
	this->timer_matmult.start();
	 get_Ax_laplace(this->g,x,K,this->eps_sqr); 
 	 this->hessmult += 1; /* there was muliplication by A */
	this->timer_matmult.stop();
	this->g -= this->b;

	/* compute function value */
	this->timer_fs.start();
 	 fx = this->get_function_value(x,true);
	 fs(all) = fx;
	this->timer_fs.stop();

	/* main cycle */
	while(this->it < maxit){
	Message("test 3");

		/* d = x - alpha_bb*g, see next step, it will be d = P(x - alpha_bb*g) - x */
		this->timer_update.start(); /* this is vector update */
		 this->d = x - alpha_bb*(this->g);
		this->timer_update.stop();

		/* d = P(d) */
		this->timer_projection.start();
		 get_projection(this->d, K);
		this->timer_projection.stop();


	Message("test 4");
		
		/* d = d - x */
		this->timer_update.start();
		 this->d -= x;
		this->timer_update.stop();

		/* Ad = A*d */
		this->timer_matmult.start();
		 get_Ax_laplace(this->Ad,this->d,K,this->eps_sqr);
		 this->hessmult += 1; /* there was multiplication by A */
		this->timer_matmult.stop();

		/* dd = dot(d,d) */
		/* dAd = dot(Ad,d) */
		/* gd = dot(g,d) */
		this->timer_dot.start();
		 dd = get_dot(this->d,this->d);
		 dAd = get_dot(this->Ad,this->d);
		 gd = get_dot(this->g,this->d);
		this->timer_dot.stop();


	Message("test 4");
		
		/* stopping criteria */
		if(dd < eps){
			break;
		}
		
		/* fx_max = max(fs) */
		this->timer_fs.start(); /* manipulation with fs */
		 fx_max = max(fs);	
		this->timer_fs.stop();
		
		/* compute step-size from A-condition */
		this->timer_stepsize.start(); /* step-size timer */
		 xi = (fx_max - fx)/dAd;
		 beta_bar = -gd/dAd;
		 beta_hat = gamma*beta_bar + sqrt(gamma*gamma*beta_bar*beta_bar + 2*xi);

		 /* beta = min(sigma2,beta_hat) */
		 if(beta_hat < sigma2){
			beta = beta_hat;
		 } else {
			beta = sigma2;
		 }
		this->timer_stepsize.stop();

	Message("test 5");


		/* update approximation and gradient */
		this->timer_update.start();/* this is vector update */
		 x += beta*(this->d); /* x = x + beta*d */
		 this->g += beta*(this->Ad); /* g = g + beta*Ad */
		this->timer_update.stop();
		
		/* compute new function value using gradient */
		this->timer_fs.start();
		 fx = this->get_function_value(x,true);
		
		 /* update fs */
		 /* fs(1:end-1) = fs(2:end); */
		 /* fs(end) = f;	*/
		 if(m == 1){
			fs(0) = fx;
		 } else {
			fs(0,m-2) = fs(1,m-1);
			fs(m-1) = fx;
		 }
		this->timer_fs.stop();

	Message("test 6");

		
		/* update BB step-size */
		this->timer_stepsize.start(); /* step-size timer */
		 alpha_bb = dd/dAd;
		this->timer_stepsize.stop();

	Message("test 7");

		
		/* print progress of algorithm */
		if(DEBUG_ALGORITHM_PRINTF || DEBUG_ALGORITHM_PRINTFS || DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[33mit = \033[0m" << this->it << std::endl;
		}

//			std::cout << "\033[36m fx = \033[0m" << fx << std::endl;

		if(DEBUG_ALGORITHM_PRINTFS){
			std::cout << "\033[36m fs = \033[0m" << fs << std::endl;
		}
		
		if(DEBUG_ALGORITHM_PRINTCOEFF){
			std::cout << "\033[36m dd = \033[0m" << dd << ",";
			std::cout << "\033[36m dAd = \033[0m" << dAd << ",";
			std::cout << "\033[36m gd = \033[0m" << gd << std::endl;
			
			std::cout << "\033[36m fx = \033[0m" << fx << ",";
			std::cout << "\033[36m fx_max = \033[0m" << fx_max << ",";
			std::cout << "\033[36m xi = \033[0m" << xi << std::endl;
			
			std::cout << "\033[36m beta_bar = \033[0m" << beta_bar << ",";
			std::cout << "\033[36m beta_hat = \033[0m" << beta_hat << ",";
			std::cout << "\033[36m beta = \033[0m" << beta << std::endl;
			
			std::cout << "\033[36m alpha_bb = \033[0m" << alpha_bb << std::endl;
			
		}
		
		/* increase iteration counter */
		this->it += 1;


	} /* main cycle end */

	this->it_all += this->it;
	this->hessmult_all += this->hessmult;

	this->timer_total.stop();

	/* say goodbye */
	if(DEBUG_ALGORITHM_SAYHELLO){
		Message_info_main("\n- final info:");
		Message_info_time(" - time: \t\t",this->timer_total.get_value_last());
		Message_info_value(" - it: \t\t\t",this->it);
		Message_info_value(" - hessmult: \t\t",this->hessmult);
		Message_info_value(" - final fx = \t\t",fx);
		Message_info("- SPGQP END ---------------------------------------------------------------");
	}

	/* very short info */
	if(DEBUG_ALGORITHM_BASIC){
		Message_info("  - SPGQP algorithm");
		Message_info_value("   - it    = ",this->it);
		Message_info_time("   - time  = ",this->timer_total.get_value_last());

	}

}

Scalar QPSolver::get_function_value(GammaVector x){
	return this->get_function_value(x,false);
}

Scalar QPSolver::get_function_value(GammaVector x, bool use_gradient){
	Scalar fx = std::numeric_limits<Scalar>::max();

	if(use_gradient){
		/* use computed gradient in this->gs to compute function value */
		GammaVector temp;
		temp = this->g;
		temp -= this->b;
		fx = 0.5*get_dot(temp,x);
	} else {
		/* we have nothing - compute fx using full formula fx = 0.5*dot(A*x,x) - dot(b,x) */
		/* for safety - do not use any allocated vector */
		
		GammaVector Ax(this->get_T()*this->get_K());
		Scalar xAx, xb;

		get_Ax_laplace(Ax,x,this->get_K(),this->eps_sqr);
		 
		xAx = get_dot(Ax,x);
		fx = 0.5*xAx;
		 
		xb = get_dot(x,this->b);
		fx -= xb;
		
	}	


	return fx;	
}


void QPSolver::print(){
	this->print(0);
}

void QPSolver::print(int nmb_of_spaces){
	int i,k;
	int K = this->get_K();
	int T = this->get_T();
	
	std::ostringstream oss_spaces;

	std::ostringstream oss;
	std::ostringstream oss_values;
	
	for(i=0;i<nmb_of_spaces;i++){
		oss_spaces << " ";
	}
	
	oss << oss_spaces.str() << "-- QP SOLVER --";
	Message_info(oss.str());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - K = ";
	Message_info_value(oss.str(),this->get_K());
	oss.str(""); oss.clear();
	
	oss << oss_spaces.str() << " - T = ";
	Message_info_value(oss.str(),this->get_T());
	oss.str(""); oss.clear();

	oss << oss_spaces.str() << " - right hand-side vector b:";
	Message_info(oss.str());
	oss.str(""); oss.clear();
	for(k=0;k<K;k++){
		oss << oss_spaces.str() << "   b[" << k << "] = ";
		oss_values << this->b(k*T,(k+1)*T-1);
		Message_info_values(oss.str(),oss_values.str());	
		oss.str(""); oss.clear();
		oss_values.str(""); oss_values.clear();
	}

}

int QPSolver::get_T(){
	return this->T;
}

int QPSolver::get_K(){
	return this->K;
}

int QPSolver::get_it(){
	return this->it;
}

int QPSolver::get_it_all(){
	return this->it_all;
}

int QPSolver::get_hessmult(){
	return this->hessmult;
}

int QPSolver::get_hessmult_all(){
	return this->hessmult_all;
}


double QPSolver::get_time_projection(){
	return this->timer_projection.get_value_sum();
}


double QPSolver::get_time_matmult(){
	return this->timer_matmult.get_value_sum();
}

double QPSolver::get_time_dot(){
	return this->timer_dot.get_value_sum();
}

double QPSolver::get_time_update(){
	return this->timer_update.get_value_sum();
}

double QPSolver::get_time_total(){
	return this->timer_total.get_value_sum();
}

double QPSolver::get_time_stepsize(){
	return this->timer_stepsize.get_value_sum();
}

double QPSolver::get_time_fs(){
	return this->timer_fs.get_value_sum();
}

