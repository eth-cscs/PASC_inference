#include "hip/hip_runtime.h"
#include "projection.h"

/* projection with timer */
void get_projection(GammaVector<Scalar> *x, int K, double *time_to_add){
	timer.start(); /* add to projection time */
	
	get_projection(x, K);
	
	(*time_to_add) += timer.stop();
}

void get_projection(GammaVector<Scalar> *x, int K){

	int t,k;
	int N = (*x).size();
	int T = N/K; /* length of vectors */
	Scalar x_sub[K];  /* GammaVector<Scalar> x_sub(K); */

#ifdef USE_GPU
	
	
#else
	#pragma omp parallel for private(t)	
	for(t=0;t<T;t++){
		/* cut x_sub from x */
		for(k=0;k<K;k++){
			x_sub[k] = (*x)(k*T+t);
		}
		
		/* compute subprojection */
		get_projection_sub(x_sub, K);

		/* add x_sub back to x */
		for(k=0;k<K;k++){
			(*x)(k*T+t) = x_sub[k];
		}
	}
#endif
}

/* project x_sub to feasible set defined by equality and inequality constraints
 * sum(x_sub) = 1
 * x_sub >= 0
 */
void get_projection_sub(Scalar *x_sub, int n){
	int i;

	bool is_inside = true;
	Scalar sum = 0.0;
	
	/* control inequality constraints */
	for(i = 0; i < n; i++){ // TODO: could be performed parallely  
		if(x_sub[i] < 0.0){
			is_inside = false;
		}
		sum += x_sub[i];
	}

	/* control equality constraints */
	if(sum != 1){ 
		is_inside = false;
	}


	/* if given point is not inside the feasible domain, then do projection */
	if(!is_inside){
		int j;
		/* compute sorted x_sub */
		Scalar y[n], sum_y;
		for(i=0;i<n;i++){
			y[i] = x_sub[i]; 
		}
		sort_bubble(y,n);

		/* now perform analytical solution of projection problem */	
		Scalar t_hat = 0.0;
		i = n - 1;
		Scalar ti;

		while(i >= 1){
			/* compute sum(y) */
			sum_y = 0.0;
			for(j=i;j<n;j++){ /* sum(y(i,n-1)) */
				sum_y += y[j];
			}
				
			ti = (sum_y - 1.0)/(Scalar)(n-i);
			if(ti >= y[i-1]){
				t_hat = ti;
				i = -1; /* break */
			} else {
				i = i - 1;
			}
		}

		if(i == 0){
			t_hat = (sum-1.0)/(Scalar)n; /* uses sum=sum(x_sub) */
		}
    
		for(i = 0; i < n; i++){ // TODO: could be performed parallely  
			/* (*x_sub)(i) = max(*x_sub-t_hat,0); */
			ti = x_sub[i] - t_hat;	
			if(ti > 0.0){
				x_sub[i] = ti;
			} else {
				x_sub[i] = 0.0;
			}
		}
	}
}

/* sort values of scalar vector */
void sort_bubble(Scalar *x, int n){
	int i;
	int m = n;
	int mnew;
	Scalar swap;

	while(m > 0){
		/* Iterate through x */
		mnew = 0;
		for(i=1;i<m;i++){
			/* Swap elements in wrong order */
			if (x[i] < x[i - 1]){
				swap = x[i];
				x[i] = x[i-1];
				x[i-1] = swap;
				mnew = i;
			}
        }
		m = mnew;
    }
}
