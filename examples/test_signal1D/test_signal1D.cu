#include "hip/hip_runtime.h"
/** @file test_test_signal1D.cu
 *  @brief test the kmeans problem solver on simple 1D signal problem
 *
 *  @author Lukas Pospisil
 */

#include "pascinference.h"
#include "solver/tssolver.h"
#include "data/signal1Ddata.h"
#include "model/graphh1fem.h"

#include <vector>

#ifndef USE_PETSCVECTOR
 #error 'This example is for PETSCVECTOR'
#endif
 
using namespace pascinference;

typedef petscvector::PetscVector PetscVector;

extern int pascinference::DEBUG_MODE;

int main( int argc, char *argv[] )
{
	/* add local program options */
	boost::program_options::options_description opt_problem("PROBLEM EXAMPLE", consoleArg.get_console_nmb_cols());
	opt_problem.add_options()
		("test_K", boost::program_options::value<int>(), "number of clusters [int]")
		("test_filename", boost::program_options::value< std::string >(), "name of input file with signal data (vector in PETSc format) [string]")
		("test_filename_out", boost::program_options::value< std::string >(), "name of output file with filtered signal data (vector in PETSc format) [string]")
		("test_filename_solution", boost::program_options::value< std::string >(), "name of input file with original signal data without noise (vector in PETSc format) [string]")
		("test_epssqr", boost::program_options::value<std::vector<double> >()->multitoken(), "penalty parameters [double]")
		("test_annealing", boost::program_options::value<int>(), "number of annealing steps [int]")
		("test_cutgamma", boost::program_options::value<bool>(), "cut gamma to set {0;1} [bool]")
		("test_scaledata", boost::program_options::value<bool>(), "scale to interval {-1,1} [bool]")
		("test_cutdata", boost::program_options::value<bool>(), "cut data to interval {0,1} [bool]")
		("test_printstats", boost::program_options::value<bool>(), "print basic statistics of data [bool]")
		("test_Theta", boost::program_options::value<std::vector<double> >()->multitoken(), "given solution Theta [K*int]")
		("test_shortinfo", boost::program_options::value<bool>(), "save shortinfo file after computation [bool]")
		("test_shortinfo_header", boost::program_options::value< std::string >(), "additional header in shortinfo [string]")
		("test_shortinfo_values", boost::program_options::value< std::string >(), "additional values in shortinfo [string]")
		("test_shortinfo_filename", boost::program_options::value< std::string >(), "name of shortinfo file [string]");

	consoleArg.get_description()->add(opt_problem);

	/* call initialize */
	if(!Initialize(argc, argv)){
		return 0;
	} 

	std::vector<double> epssqr_list;
	if(consoleArg.set_option_value("test_epssqr", &epssqr_list)){
		/* sort list */
		std::sort(epssqr_list.begin(), epssqr_list.end(), std::less<double>());
		
	} else {
		std::cout << "test_epssqr has to be set! Call application with parameter -h to see all parameters" << std::endl;
		return 0;
	}

	int K, annealing; 
	double graph_coeff; 
	bool cutgamma, scaledata, cutdata, printstats, shortinfo_write_or_not;

	std::string filename;
	std::string filename_out;
	std::string filename_solution;
	std::string shortinfo_filename;
	std::string shortinfo_header;
	std::string shortinfo_values;

	consoleArg.set_option_value("test_K", &K, 2);
	consoleArg.set_option_value("test_filename", &filename, "data/samplesignal.bin");
	consoleArg.set_option_value("test_filename_out", &filename_out, "samplesignal");
	consoleArg.set_option_value("test_filename_solution", &filename_solution, "");
	consoleArg.set_option_value("test_annealing", &annealing, 1);
	consoleArg.set_option_value("test_cutgamma", &cutgamma, false);
	consoleArg.set_option_value("test_scaledata", &scaledata, false);
	consoleArg.set_option_value("test_cutdata", &cutdata, true);
	consoleArg.set_option_value("test_printstats", &printstats, false);
	consoleArg.set_option_value("test_shortinfo", &shortinfo_write_or_not, true);
	consoleArg.set_option_value("test_shortinfo_header", &shortinfo_header, "");
	consoleArg.set_option_value("test_shortinfo_values", &shortinfo_values, "");
	consoleArg.set_option_value("test_shortinfo_filename", &shortinfo_filename, "shortinfo/samplesignal.txt");

	/* maybe theta is given in console parameters */
	bool given_Theta;
	std::vector<double> Theta_list;
	double Theta_solution[K];
	if(consoleArg.set_option_value("test_Theta", &Theta_list)){
		given_Theta = true;
		
		/* control number of provided Theta */
		if(Theta_list.size() != K){
			coutMaster << "number of provided Theta solutions is different then number of clusters!" << std::endl;
			return 0;
		}

		/* store solution in array */
		for(int k=0;k < K;k++){
			Theta_solution[k] = Theta_list[k];
		}
	} else {
		given_Theta = false;
	}	

	/* set decomposition in space */
	int DDT_size = GlobalManager.get_size();

	coutMaster << "- PROBLEM INFO ----------------------------" << std::endl;
	coutMaster << " DDT_size                    = " << std::setw(30) << DDT_size << " (decomposition in space)" << std::endl;
	coutMaster << " test_K                      = " << std::setw(30) << K << " (number of clusters)" << std::endl;
	if(given_Theta){
		coutMaster << " test_Theta                  = " << std::setw(30) << print_array(Theta_solution,K) << std::endl;
	}

	coutMaster << " test_filename               = " << std::setw(30) << filename << " (name of input file with signal data)" << std::endl;
	coutMaster << " test_filename_out           = " << std::setw(30) << filename_out << " (name of output file with filtered signal data)" << std::endl;
	coutMaster << " test_filename_solution      = " << std::setw(30) << filename_solution << " (name of input file with original signal data without noise)" << std::endl;
	coutMaster << " test_epssqr                 = " << std::setw(30) << print_vector(epssqr_list) << " (penalty parameters)" << std::endl;
	coutMaster << " test_annealing              = " << std::setw(30) << annealing << " (number of annealing steps)" << std::endl;
	coutMaster << " test_cutgamma               = " << std::setw(30) << cutgamma << " (cut gamma to {0;1})" << std::endl;
	coutMaster << " test_cutdata                = " << std::setw(30) << cutdata << " (cut data to {0,1})" << std::endl;
	coutMaster << " test_scaledata              = " << std::setw(30) << scaledata << " (scale data to {-1,1})" << std::endl;
	coutMaster << " test_printstats             = " << std::setw(30) << printstats << " (print basic statistics of data)" << std::endl;
	coutMaster << " test_shortinfo              = " << std::setw(30) << shortinfo_write_or_not << " (save shortinfo file after computation)" << std::endl;
	coutMaster << " test_shortinfo_header       = " << std::setw(30) << shortinfo_header << " (additional header in shortinfo)" << std::endl;
	coutMaster << " test_shortinfo_values       = " << std::setw(30) << shortinfo_values << " (additional values in shortinfo)" << std::endl;
	coutMaster << " test_shortinfo_filename     = " << std::setw(30) << shortinfo_filename << " (name of shortinfo file)" << std::endl;
	coutMaster << "-------------------------------------------" << std::endl << std::endl;


	/* start logging */
	std::ostringstream oss;
	oss << "log/" << filename_out << ".txt";
	logging.begin(oss.str());
	oss.str("");

	/* start shortinfo output */
	if(shortinfo_write_or_not){
		shortinfo.begin(shortinfo_filename);
	}
	std::ostringstream oss_short_output_values;
	std::ostringstream oss_short_output_header;
		
	/* say hello */
	coutMaster << "- start program" << std::endl;

/* 1.) prepare preliminary time-series data (to get the size of the problem T) */
	coutMaster << "--- PREPARING PRELIMINARY DATA ---" << std::endl;
	Signal1DData<PetscVector> mydata(filename);

/* 2.) prepare decomposition */
	coutMaster << "--- COMPUTING DECOMPOSITION ---" << std::endl;

	/* prepare decomposition based on preloaded data */
	Decomposition decomposition(mydata.get_Tpreliminary(), 1, K, 1, DDT_size);

	/* print info about decomposition */
	decomposition.print(coutMaster);

/* 3.) prepare time-series data */
	coutMaster << "--- APPLY DECOMPOSITION TO DATA ---" << std::endl;

	mydata.set_decomposition(decomposition);

	/* print information about loaded data */
	mydata.print(coutMaster);

	/* print statistics */
	if(printstats) mydata.printstats(coutMaster);

	/* cut data */
	if(cutdata) mydata.cutdata(0,1);

	/* scale data */
	if(scaledata) mydata.scaledata(-1,1,0,1);

/* 4.) prepare model */
	coutMaster << "--- PREPARING MODEL ---" << std::endl;

	/* prepare model on the top of given data */
	GraphH1FEMModel<PetscVector> mymodel(mydata, epssqr_list[0]);

	/* print info about model */
	mymodel.print(coutMaster,coutAll);

/* 5.) prepare time-series solver */
	coutMaster << "--- PREPARING SOLVER ---" << std::endl;

	/* prepare time-series solver */
	TSSolver<PetscVector> mysolver(mydata, annealing);

	/* print info about solver */
	mysolver.print(coutMaster,coutAll);

	/* set solution if obtained from console */
	if(given_Theta)	mysolver.set_solution_theta(Theta_solution);
	
/* 6.) solve the problem with initial epssqr */
	coutMaster << "--- SOLVING THE PROBLEM with epssqr = " << epssqr_list[0] << " ---" << std::endl;
	mysolver.solve();

	/* cut gamma */
	if(cutgamma) mydata.cutgamma();

	/* unscale data before save */
	if(scaledata) mydata.scaledata(0,1,-1,1);

	coutMaster << "--- SAVING OUTPUT ---" << std::endl;
	oss << filename_out << "_depth0" << "_epssqr" << epssqr_list[0];
//	mydata.saveImage(oss.str(),true);
	oss.str("");

	/* write short output */
	if(shortinfo_write_or_not){

		/* add provided strings from console parameters and info about the problem */
		oss_short_output_header << shortinfo_header << "filename,K,depth,epssqr,";
		oss_short_output_values << shortinfo_values << filename << "," << K << ",0,0.0,"; 

		/* append Theta solution */
		for(int k=0; k<K; k++) oss_short_output_header << "Theta" << k << ",";
		oss_short_output_values << mydata.print_thetavector(); 

		/* print info from solver */
		mysolver.printshort(oss_short_output_header, oss_short_output_values);

		/* append end of line */
		oss_short_output_header << std::endl;
		oss_short_output_values << std::endl;

		/* write to shortinfo file */
		shortinfo.write(oss_short_output_header.str());
		shortinfo.write(oss_short_output_values.str());
			
		/* clear streams for next writing */
		oss_short_output_header.str("");
		oss_short_output_values.str("");
	
	}


/* 7.) solve the problems with other epssqr */
	for(int depth = 1; depth < epssqr_list.size();depth++){
		/* set new epssqr */
		mymodel.set_epssqr(epssqr_list[depth]);

		/* decrease the number of annealing steps in TSSolver to 1 */
		mysolver.set_annealing(1);

		/* scale data before computation */
		if(scaledata) mydata.scaledata(-1,1,0,1);

		coutMaster << "--- SOLVING THE PROBLEM with epssqr = " << epssqr_list[depth] << " ---" << std::endl;
		mysolver.solve();

		/* cut gamma */
		if(cutgamma) mydata.cutgamma();

		/* unscale data before export */
		if(scaledata) mydata.scaledata(0,1,-1,1);

		coutMaster << "--- SAVING OUTPUT ---" << std::endl;
		oss << image_out << "_depth" << depth << "_epssqr" << epssqr_list[depth];
//		mydata.saveImage(oss.str(),false);
		oss.str("");
		
		/* write short output */
		if(shortinfo_write_or_not){
			/* add provided strings from console parameters */
			oss_short_output_values << shortinfo_values << filename << "," << K << "," << depth << "," << epssqr_list[depth] << ",";

			/* append Theta solution */
			oss_short_output_values << mydata.print_thetavector(); 

			/* append data from solver */
			mysolver.printshort(oss_short_output_header, oss_short_output_values);

			/* append end of line */
			oss_short_output_values << std::endl;

			/* write data */
			shortinfo.write(oss_short_output_values.str());

			/* clear streams for next time */
			oss_short_output_values.str("");
		}
	}

	/* print solution */
	coutMaster << "--- THETA SOLUTION ---" << std::endl;
	mydata.print_thetavector(coutMaster);

	/* print timers */
	coutMaster << "--- TIMERS INFO ---" << std::endl;
	mysolver.printtimer(coutMaster);

	/* print short info */
	coutMaster << "--- FINAL SOLVER INFO ---" << std::endl;
	mysolver.printstatus(coutMaster);

	/* say bye */	
	coutMaster << "- end program" << std::endl;

	logging.end();
	Finalize();

	return 0;
}

